#include "hip/hip_runtime.h"
#include <dnn-utility.h>

map<int, int> getLabelMapping(const hmat& labels) {
  map<int, int> classes;
  for (size_t i=0; i<labels.size(); ++i)
    classes[(int) labels[i]] = 1;

  int counter = 0;
  map<int, int>::iterator itr = classes.begin();
  for (; itr != classes.end(); ++itr)
    itr->second = ++counter;

  return classes;
}

namespace ext {

  void rescale(mat& data, float lower, float upper) {
    float min = ext::min(data);
    float max = ext::max(data);

    float ratio = (upper - lower) / (max - min);
    data = (data - min) * ratio + lower;
  }

  float max(const mat& v) {
    thrust::device_ptr<float> vPtr(v.getData());
    thrust::device_ptr<float> maxPtr = thrust::max_element(vPtr, vPtr + v.size());
    thrust::host_vector<float> hMaxPtr(maxPtr, maxPtr + 1);
    return hMaxPtr[0];
  }

  float min(const mat& v) {
    thrust::device_ptr<float> vPtr(v.getData());
    thrust::device_ptr<float> minPtr = thrust::min_element(vPtr, vPtr + v.size());
    thrust::host_vector<float> hMaxPtr(minPtr, minPtr + 1);
    return hMaxPtr[0];
  }

  float max(const hmat& v) {
    float* m = thrust::max_element(v.getData(), v.getData() + v.size());
    return *m;
  }

  float min(const hmat& v) {
    float* m = thrust::min_element(v.getData(), v.getData() + v.size());
    return *m;
  }
};

mat getError(const mat& target, const mat& output, ERROR_MEASURE errorMeasure) {

  mat error;

  const mat& O = output;

  switch (errorMeasure) {
    case L2ERROR: 
      error = output - target;
      error.reserve(error.getRows() * (error.getCols() + 1));
      error.resize(error.getRows(), error.getCols() + 1);

      break;
    case CROSS_ENTROPY: {

	size_t output_dim = target.getCols();

	error.resize(target.getRows(), target.getCols() + 1);

	thrust::device_ptr<float> pPtr(target.getData());
	thrust::device_ptr<float> oPtr(O.getData());

	thrust::device_ptr<float> ePtr(error.getData());

	thrust::device_vector<float> TMP(O.size());
	thrust::transform(oPtr, oPtr + O.size(), TMP.begin(), func::min_threshold<float>(1e-10));

	thrust::transform(pPtr, pPtr + target.size(), TMP.begin(), ePtr, func::dcrossentropy<float>());

	break;
      }

    default:
      break;
  }

  return error;
}

mat posteriorProb2Label(const mat& prob) {

  assert(prob.getCols() > 1);

  size_t rows = prob.getRows(),
	 cols = prob.getCols();

  float* h_prob = new float[prob.size()];
  float* h_labels  = new float[rows];
  CCE(hipMemcpy(h_prob, prob.getData(), sizeof(float) * prob.size(), hipMemcpyDeviceToHost));

  for (size_t i=0; i<rows; ++i) {

    float max = -1e10;
    size_t maxIdx = 0;

    for (size_t j=0; j<cols; ++j) {
      if (h_prob[j * rows + i] > max) {
	max = h_prob[j * rows + i];
	maxIdx = j;
      }
    }

    h_labels[i] = maxIdx + 1;
  }

  mat labels(h_labels, rows, 1);

  delete [] h_prob;
  delete [] h_labels;

  return labels;
}

vector<float> copyToHost(const mat& m) {
  vector<float> hm(m.size());
  thrust::device_ptr<float> dPtr(m.getData());
  thrust::copy(dPtr, dPtr + m.size(), hm.begin());
  return hm;
}

size_t countDifference(const mat& m1, const mat& m2) {
  assert(m1.size() == m2.size());

  size_t L = m1.size();
  thrust::device_ptr<float> ptr1(m1.getData());
  thrust::device_ptr<float> ptr2(m2.getData());

  size_t nDiff = thrust::inner_product(ptr1, ptr1 + L, ptr2, 0.0, thrust::plus<float>(), thrust::not_equal_to<float>());
  return nDiff;
}


size_t zeroOneError(const mat& prob, const mat& label, ERROR_MEASURE errorMeasure) {
  assert(prob.getRows() == label.getRows());
  assert(label.getCols() == 1);

  size_t nError = 0;

  if (errorMeasure == L2ERROR) {
    // nError = countDifference(label, prob);
  }
  else {
    mat L = posteriorProb2Label(prob);
    nError = countDifference(L, label);
  }

  return nError;
}

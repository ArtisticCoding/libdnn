#include <dnn.h>
#include <thrust/extrema.h>

DNN::DNN(): _transforms(), _config() {}

DNN::DNN(string fn): _transforms(), _config() {
  this->read(fn);
}

DNN::DNN(const Config& config): _transforms(), _config(config) {
}

DNN::DNN(const DNN& source): _transforms(source._transforms.size()), _config() {

  for (size_t i=0; i<_transforms.size(); ++i)
    _transforms[i] = source._transforms[i]->clone();
}

void DNN::init(const std::vector<mat>& weights) {
  _transforms.resize(weights.size());

  for (size_t i=0; i<_transforms.size(); ++i)
      _transforms[i] = new Sigmoid(weights[i]);
  _transforms.back() = new Softmax(weights.back());
}

void DNN::init(const std::vector<size_t>& dims) {
  assert(dims.size() > 0);
  size_t L = dims.size() - 1;

  _transforms.resize(L);

  for (size_t i=0; i<L; ++i) {
    size_t M = dims[i] + 1;
    size_t N = dims[i+1] + 1;

    if (i == L-1)
      _transforms[i] = new Softmax(M, N, _config.variance);
    else
      _transforms[i] = new Sigmoid(M, N, _config.variance);
  }
}

DNN::~DNN() {
  for (size_t i=0; i<_transforms.size(); ++i)
    delete _transforms[i];
}

DNN& DNN::operator = (DNN rhs) {
  swap(*this, rhs);
  return *this;
}
  
void DNN::setConfig(const Config& config) {
  _config = config;
}

size_t DNN::getNLayer() const {
  return _transforms.size() + 1;
}

#pragma GCC diagnostic ignored "-Wunused-result"
void readweight(FILE* fid, float* w, size_t rows, size_t cols) {

  for (size_t i=0; i<rows - 1; ++i)
    for (size_t j=0; j<cols; ++j)
      fscanf(fid, "%f ", &(w[j * rows + i]));

  fscanf(fid, "]\n<bias>\n [");

  for (size_t j=0; j<cols; ++j)
    fscanf(fid, "%f ", &(w[j * rows + rows - 1]));
  fscanf(fid, "]\n");

}

void DNN::read(string fn) {
  FILE* fid = fopen(fn.c_str(), "r");

  _transforms.clear();

  size_t rows, cols;
  char type[80];

  while (fscanf(fid, "%s", type) != EOF) {
    fscanf(fid, "%lu %lu\n [\n", &rows, &cols);
    printf("\33[34m%-17s\33[0m %-6lu x %-6lu \n", type, rows, cols);

    float* hw = new float[(rows + 1) * (cols + 1)];
    readweight(fid, hw, rows + 1, cols);

    // Reserve one more column for bias)
    mat w(hw, rows + 1, cols + 1);

    string transformType = string(type);
    if (transformType == "<sigmoid>")
      _transforms.push_back(new Sigmoid(w));
    else if (transformType == "<softmax>")
      _transforms.push_back(new Softmax(w));

    delete [] hw;
  }

  fclose(fid);
}

void DNN::save(string fn) const {
  FILE* fid = fopen(fn.c_str(), "w");

  for (size_t i=0; i<_transforms.size(); ++i) {
    const mat& w = _transforms[i]->getW();

    size_t rows = w.getRows();
    size_t cols = w.getCols() - 1;

    fprintf(fid, "<%s> %lu %lu \n", _transforms[i]->toString().c_str(), rows - 1, cols);
    fprintf(fid, " [");

    // ==============================
    std::vector<float> data = copyToHost(w);
    // float* data = new float[w.size()];
    // CCE(hipMemcpy(data, w.getData(), sizeof(float) * w.size(), hipMemcpyDeviceToHost));

    for (size_t j=0; j<rows-1; ++j) {
      fprintf(fid, "\n  ");
      for (size_t k=0; k<cols; ++k)
	fprintf(fid, "%g ", data[k * rows + j]);
    }
    fprintf(fid, "]\n");

    fprintf(fid, "<bias> \n [");
    for (size_t j=0; j<cols; ++j)
      fprintf(fid, "%g ", data[j * rows + rows - 1]);
    fprintf(fid, " ]\n");

    // delete [] data;
  }

  printf("nn_structure ");
  for (size_t i=0; i<_transforms.size(); ++i)
    printf("%lu ", _transforms[i]->getW().getRows());
  printf("%lu\n", _transforms.back()->getW().getCols());
  
  fclose(fid);
}

void DNN::print() const {
  for (size_t i=0; i<_transforms.size(); ++i)
    _transforms[i]->getW().print();
}

// ========================
// ===== Feed Forward =====
// ========================

void print(const thrust::host_vector<float>& hv) {
  cout << "\33[33m[";
  for (size_t i=0; i<hv.size(); ++i)
    cout << hv[i] << " ";
  cout << " ] \33[0m" << endl << endl;
}

void print(const thrust::device_vector<float>& dv) {
  thrust::host_vector<float> hv(dv.begin(), dv.end());
  ::print(hv);
}

void DNN::adjustLearningRate(float trainAcc) {
  static size_t phase = 0;

  if ( (trainAcc > 0.80 && phase == 0) ||
       (trainAcc > 0.85 && phase == 1) ||
       (trainAcc > 0.90 && phase == 2) ||
       (trainAcc > 0.92 && phase == 3) ||
       (trainAcc > 0.95 && phase == 4) ||
       (trainAcc > 0.97 && phase == 5)
     ) {

    float ratio = 0.9;
    printf("\33[33m[Info]\33[0m Adjust learning rate from \33[32m%.7f\33[0m to \33[32m%.7f\33[0m\n", _config.learningRate, _config.learningRate * ratio);
    _config.learningRate *= ratio;
    ++phase;
  }
}

void DNN::feedForward(mat& output, const mat& fin) {

  // FIXME This should be an ASSERTION, not resizing.
  if (_houts.size() != this->getNLayer() - 2)
    _houts.resize(this->getNLayer() - 2);

  _transforms[0]->feedForward(_houts[0], fin);

  for (size_t i=1; i<_transforms.size()-1; ++i)
    _transforms[i]->feedForward(_houts[i], _houts[i-1]);

  _transforms.back()->feedForward(output, _houts.back());

  output.resize(output.getRows(), output.getCols() - 1);
}

// ============================
// ===== Back Propagation =====
// ============================

void DNN::backPropagate(mat& error, const mat& fin, const mat& fout) {
  _transforms.back()->backPropagate(error, _houts.back(), fout);

  for (int i=_transforms.size() - 2; i >= 1; --i)
    _transforms[i]->backPropagate(error, _houts[i-1], _houts[i]);

  _transforms[0]->backPropagate(error, fin, _houts[0]);
}

void DNN::update(float learning_rate) { 
  for (size_t i=0; i<_transforms.size(); ++i)
    _transforms[i]->update(learning_rate);
}

Config DNN::getConfig() const {
  return _config;
}

void swap(DNN& lhs, DNN& rhs) {
  using WHERE::swap;
  swap(lhs._transforms, rhs._transforms);
  swap(lhs._config, rhs._config);
}

// =============================
// ===== Utility Functions =====
// =============================

/*mat l2error(mat& targets, mat& predicts) {
  mat err(targets - predicts);

  thrust::device_ptr<float> ptr(err.getData());
  thrust::transform(ptr, ptr + err.size(), ptr, func::square<float>());

  mat sum_matrix(err.getCols(), 1);
  err *= sum_matrix;
  
  return err;
}
*/

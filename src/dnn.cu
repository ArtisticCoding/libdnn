#include "hip/hip_runtime.h"
#include <dnn.h>
#include <dnn-utility.h>
#include <thrust/extrema.h>

DNN::DNN() {}

DNN::DNN(string fn): _dims(0) {
  this->read(fn);
}

DNN::DNN(const std::vector<size_t>& dims): _dims(dims) {
  size_t L = _dims.size() - 1;

  _transforms.resize(L);

  for (size_t i=0; i<L; ++i) {
    size_t M = _dims[i] + 1;
    size_t N = _dims[i+1] + 1;

    if (i == L-1) {
      _transforms[i] = Softmax(M, N);
      cout << "toString() = " <<  _transforms[i].toString() << endl;
    }
    else
      _transforms[i] = AffineTransform(M, N);
  }
}

DNN::DNN(const DNN& source): 
  _dims(source._dims),
  _transforms(source._transforms) {
}

DNN& DNN::operator = (DNN rhs) {
  swap(*this, rhs);
  return *this;
}

size_t DNN::getNLayer() const {
  return _dims.size(); 
}

size_t DNN::getDepth() const {
  return _dims.size() - 2;
}

#pragma GCC diagnostic ignored "-Wunused-result"
void readweight(FILE* fid, float* w, size_t rows, size_t cols) {

  for (size_t i=0; i<rows - 1; ++i)
    for (size_t j=0; j<cols; ++j)
      fscanf(fid, "%f ", &(w[j * rows + i]));

  fscanf(fid, "]\n<sigmoid>\n [");

  for (size_t j=0; j<cols; ++j)
    fscanf(fid, "%f ", &(w[j * rows + rows - 1]));
  fscanf(fid, "]\n");

}

#pragma GCC diagnostic ignored "-Wunused-result"
void DNN::read(string fn) {
  FILE* fid = fopen(fn.c_str(), "r");

  _dims.clear();
  _transforms.clear();

  size_t rows, cols;
  char type[80];

  while (fscanf(fid, "%s", type) != EOF) {
    fscanf(fid, "%lu %lu\n [\n", &rows, &cols);
    printf("%s: rows = %lu, cols = %lu \n", type, rows, cols);

    float* hw = new float[(rows + 1) * (cols + 1)];
    readweight(fid, hw, rows + 1, cols);

    // Reserve one more column for bias)
    mat w(hw, rows + 1, cols + 1);

    string transformType = string(type);
    if (transformType == "<affinetransform>")
      _transforms.push_back(AffineTransform(w));
    else if (transformType == "<softmax>")
      _transforms.push_back(Softmax(w));

    delete [] hw;

    _dims.push_back(rows);
  }
  _dims.push_back(cols);

  fclose(fid);
}

void DNN::save(string fn) const {
  FILE* fid = fopen(fn.c_str(), "w");

  for (size_t i=0; i<_transforms.size(); ++i) {
    const mat& w = _transforms[i].getW();

    size_t rows = w.getRows();
    size_t cols = w.getCols() - 1;

    fprintf(fid, "<%s> %lu %lu \n", _transforms[i].toString().c_str(), rows - 1, cols);
    fprintf(fid, " [");

    // ==============================
    float* data = new float[w.size()];
    CCE(hipMemcpy(data, w.getData(), sizeof(float) * w.size(), hipMemcpyDeviceToHost));

    for (size_t j=0; j<rows-1; ++j) {
      fprintf(fid, "\n  ");
      for (size_t k=0; k<cols; ++k)
	fprintf(fid, "%g ", data[k * rows + j]);
    }
    fprintf(fid, "]\n");

    fprintf(fid, "<sigmoid> \n [");
    for (size_t j=0; j<cols; ++j)
      fprintf(fid, "%g ", data[j * rows + rows - 1]);
    fprintf(fid, " ]\n");

    delete [] data;
  }

  fprintf(stdout, "nn_structure ");
  for (size_t i=0; i<_dims.size(); ++i)
    fprintf(stdout, "%lu ", _dims[i]);
  fprintf(stdout, "\n");
  
  fclose(fid);
}

void DNN::print() const {
  for (size_t i=0; i<_transforms.size(); ++i)
    _transforms[i].getW().print(stdout);
}

// ========================
// ===== Feed Forward =====
// ========================

void print(const thrust::host_vector<float>& hv) {
  cout << "\33[33m[";
  for (size_t i=0; i<hv.size(); ++i)
    cout << hv[i] << " ";
  cout << " ] \33[0m" << endl << endl;
}

void print(const mat& m) {
  thrust::device_ptr<float> dm(m.getData());
  thrust::host_vector<float> hm(dm, dm + m.size());

  ::print(hm);
}

void print(const thrust::device_vector<float>& dv) {
  thrust::host_vector<float> hv(dv.begin(), dv.end());
  ::print(hv);
}

void DNN::train(const DataSet& train, const DataSet& valid, size_t batchSize, ERROR_MEASURE errorMeasure) {

  printf("Training...\n");
  perf::Timer timer;
  timer.start();

  vector<mat> O(this->getNLayer());

  size_t input_dim = train.X.getCols(),
	 output_dim= train.y.getCols();

  size_t Ein, Eout;
  size_t prevEout = valid.y.size();
  size_t MAX_EPOCH = 1024, epoch;

  size_t nTrain = train.X.getRows(),
	 nValid = valid.X.getRows();

  size_t nBatch = nTrain / batchSize,
         remained = nTrain - nBatch * batchSize;

  if (remained > 0)
    ++nBatch;

  for (epoch=0; epoch<MAX_EPOCH; ++epoch) {

    for (size_t b=0; b<nBatch; ++b) {

      size_t offset = b*batchSize;
      size_t nData = batchSize;

      if (b == nBatch - 1)
	nData = min(remained - 1, batchSize);

      this->feedForward(train, O, offset, nData);

      mat error = this->getError(train.y, O.back(), offset, nData, errorMeasure);

      this->backPropagate(train, O, error, offset, nData);
      this->updateParameters(1e-1);
    }

    this->feedForward(valid, O);

    Eout = zeroOneError(O.back(), valid.y, errorMeasure);

    if (Eout > prevEout && (float) Eout / nValid < 0.2)
      break;

    prevEout = Eout;
  }

  // Show Summary
  printf("\n%d epochs in total\n", epoch);
  timer.elapsed();

  this->feedForward(train, O);
  Ein = zeroOneError(O.back(), train.y, errorMeasure);

  printf("[   In-Sample   ] ");
  showAccuracy(Ein, train.y.size());
  printf("[ Out-of-Sample ] ");
  showAccuracy(Eout, valid.y.size());
}

mat DNN::getError(const mat& target, const mat& output, size_t offset, size_t batchSize, ERROR_MEASURE errorMeasure) {

  mat error;

  mat& O = const_cast<mat&>(output);

  switch (errorMeasure) {
    case L2ERROR: 
      // mat error = O.back() - train.y;
      error = calcError(O, target, offset, batchSize);
      error.reserve(error.getRows() * (error.getCols() + 1));
      error.resize(error.getRows(), error.getCols() + 1);

      break;
    case CROSS_ENTROPY: {

	error.resize(batchSize, target.getCols() + 1);

	mat partialTarget(batchSize, target.getCols() + 1);
	memcpy2D(partialTarget, target, offset, 0, batchSize, target.getCols(), 0, 0);

	thrust::device_ptr<float> pPtr(partialTarget.getData());
	thrust::device_ptr<float> oPtr(O.getData());

	thrust::device_ptr<float> ePtr(error.getData());

	thrust::transform(pPtr, pPtr + partialTarget.size(), oPtr, ePtr, func::dcrossentropy<float>());

	break;
      }

    default:
      break;
  }

  O.resize(O.getRows(), O.getCols() + 1);

  return error;
}

void DNN::feedForward(const DataSet& data, std::vector<mat>& O, size_t offset, size_t batchSize) {
  assert(batchSize >= 0 && offset + batchSize <= data.X.getRows());

  // All data in one-batch (Gradient Descent)
  if (batchSize == 0)
    batchSize = data.X.getRows();

  assert(O.size() == _dims.size());

  O[0].resize(batchSize, data.X.getCols());
  memcpy2D(O[0], data.X, offset, 0, batchSize, data.X.getCols(), 0, 0);

  for (size_t i=0; i<_transforms.size(); ++i)
    _transforms[i].feedForward(O[i+1], O[i], offset, batchSize);

  O.back().resize(O.back().getRows(), O.back().getCols() - 1);
}

// ============================
// ===== Back Propagation =====
// ============================

void DNN::backPropagate(const DataSet& data, std::vector<mat>& O, mat& error, size_t offset, size_t batchSize) {

  for (int i=_transforms.size() - 1; i >= 0; --i)
    _transforms[i].backPropagate(O[i], O[i+1], error);
}

void DNN::updateParameters(float learning_rate) { 
  for (size_t i=0; i<_transforms.size(); ++i)
    _transforms[i].update(learning_rate);
}

void swap(DNN& lhs, DNN& rhs) {
  using WHERE::swap;
  swap(lhs._dims, rhs._dims);
  swap(lhs._transforms, rhs._transforms);
}

// =============================
// ===== Utility Functions =====
// =============================

mat l2error(mat& targets, mat& predicts) {
  mat err(targets - predicts);

  thrust::device_ptr<float> ptr(err.getData());
  thrust::transform(ptr, ptr + err.size(), ptr, func::square<float>());

  mat sum_matrix(err.getCols(), 1);
  err *= sum_matrix;
  
  return err;
}


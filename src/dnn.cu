#include "hip/hip_runtime.h"
#include <dnn.h>
#include <dnn-utility.h>

DNN::DNN() {}

DNN::DNN(string fn): _dims(0) {
  this->read(fn);
}

DNN::DNN(const std::vector<size_t>& dims): _dims(dims) {
  size_t L = _dims.size() - 1;

  _transforms.resize(L);

  for (size_t i=0; i<L; ++i) {
    size_t M = _dims[i] + 1;
    size_t N = _dims[i+1];

    // If not output layer, reserve last column for bias 
    if (i < L - 1)
      N += 1;

    _transforms[i] = AffineTransform(M, N);
  }
}

DNN::DNN(const DNN& source): 
  _dims(source._dims),
  _transforms(source._transforms) {
}

DNN& DNN::operator = (DNN rhs) {
  swap(*this, rhs);
  return *this;
}

size_t DNN::getNLayer() const {
  return _dims.size(); 
}

size_t DNN::getDepth() const {
  return _dims.size() - 2;
}

#pragma GCC diagnostic ignored "-Wunused-result"
void readweight(FILE* fid, float* w, size_t rows, size_t cols) {

  for (size_t i=0; i<rows - 1; ++i)
    for (size_t j=0; j<cols; ++j)
      fscanf(fid, "%f ", &(w[j * rows + i]));

  fscanf(fid, "]\n<sigmoid>\n [");

  for (size_t j=0; j<cols; ++j)
    fscanf(fid, "%f ", &(w[j * rows + rows - 1]));
  fscanf(fid, "]\n");

}

#pragma GCC diagnostic ignored "-Wunused-result"
void DNN::read(string fn) {
  FILE* fid = fopen(fn.c_str(), "r");

  _dims.clear();
  _transforms.clear();

  size_t rows, cols;

  while (fscanf(fid, "<affinetransform> %lu %lu\n [\n", &rows, &cols) != EOF) {

    printf("rows = %lu, cols = %lu \n", rows, cols);

    float* hw = new float[(rows + 1) * cols];
    readweight(fid, hw, rows + 1, cols);

    // Reserve one more column for bias)
    mat w(rows + 1, cols + 1);
    CCE(hipMemcpy(w.getData(), hw, sizeof(float) * (rows + 1) * cols, hipMemcpyHostToDevice));
    _transforms.push_back(AffineTransform(w));
    delete [] hw;

    _dims.push_back(rows);
  }
  _dims.push_back(cols);

  // No need for one more column in the last weight matrix, resize it back.
  // (since I cannot tell which "i" is the last one in the while loop. )
  _transforms.back().resize(rows + 1, cols);
  
  fclose(fid);
}

void DNN::save(string fn) const {
  FILE* fid = fopen(fn.c_str(), "w");

  for (size_t i=0; i<_transforms.size(); ++i) {
    const mat& w = _transforms[i].getW();

    size_t rows = w.getRows();
    size_t cols = w.getCols();

    if (i != _transforms.size() - 1)
      cols -= 1;

    fprintf(fid, "<affinetransform> %lu %lu \n", rows - 1, cols);
    fprintf(fid, " [");

    // ==============================
    float* data = new float[w.size()];
    CCE(hipMemcpy(data, w.getData(), sizeof(float) * w.size(), hipMemcpyDeviceToHost));

    for (size_t j=0; j<rows-1; ++j) {
      fprintf(fid, "\n  ");
      for (size_t k=0; k<cols; ++k)
	fprintf(fid, "%g ", data[k * rows + j]);
    }
    fprintf(fid, "]\n");

    fprintf(fid, "<sigmoid> \n [");
    for (size_t j=0; j<cols; ++j)
      fprintf(fid, "%g ", data[j * rows + rows - 1]);
    fprintf(fid, " ]\n");

    delete [] data;
  }

  fprintf(stdout, "nn_structure ");
  for (size_t i=0; i<_dims.size(); ++i)
    fprintf(stdout, "%lu ", _dims[i]);
  fprintf(stdout, "\n");
  
  fclose(fid);
}

void DNN::print() const {
  for (size_t i=0; i<_transforms.size(); ++i)
    _transforms[i].getW().print(stdout);
}

// ========================
// ===== Feed Forward =====
// ========================

void print(const thrust::host_vector<float>& hv) {
  cout << "\33[33m[";
  for (size_t i=0; i<hv.size(); ++i)
    cout << hv[i] << " ";
  cout << " ] \33[0m" << endl << endl;
}

void print(const mat& m) {
  thrust::device_ptr<float> dm(m.getData());
  thrust::host_vector<float> hm(dm, dm + m.size());

  ::print(hm);
}

void print(const thrust::device_vector<float>& dv) {
  thrust::host_vector<float> hv(dv.begin(), dv.end());
  ::print(hv);
}

void DNN::train(const DataSet& train, const DataSet& valid, size_t batchSize, ERROR_MEASURE err) {

  printf("Training...\n");
  perf::Timer timer;
  timer.start();

  vector<mat> O(this->getNLayer());

  size_t input_dim = train.X.getCols(),
	 output_dim= train.y.getCols();

  size_t Ein, Eout;
  size_t prevEout = valid.y.size();
  size_t MAX_EPOCH = 1024, epoch;

  size_t nTrain = train.X.getRows(),
	 nValid = valid.X.getRows();

  size_t nBatch = nTrain / batchSize,
         remained = nTrain - nBatch * batchSize;

  if (remained > 0)
    ++nBatch;

  for (epoch=0; epoch<MAX_EPOCH; ++epoch) {

    for (size_t b=0; b<nBatch; ++b) {

      size_t offset = b*batchSize;
      size_t nData = batchSize;

      if (b == nBatch - 1)
	nData = min(remained - 1, batchSize);

      this->feedForward(train, O, offset, nData);
      this->backPropagate(train, O, offset, nData);
      this->updateParameters(5 * 1e-3);
    }

    this->feedForward(valid, O);

    Eout = zeroOneError(O.back(), valid.y);

    if (Eout > prevEout && (float) Eout / nValid < 0.2)
      break;

    prevEout = Eout;
  }

  // Show Summary
  printf("\n%d epochs in total\n", epoch);
  timer.elapsed();

  this->feedForward(train, O);
  Ein = zeroOneError(O.back(), train.y);

  printf("[   In-Sample   ] ");
  showAccuracy(Ein, train.y.size());
  printf("[ Out-of-Sample ] ");
  showAccuracy(Eout, valid.y.size());

}

void DNN::feedForward(const DataSet& data, std::vector<mat>& O, size_t offset, size_t batchSize) {
  assert(batchSize >= 0 && offset + batchSize <= data.X.getRows());

  // All data in one-batch (Gradient Descent)
  if (batchSize == 0)
    batchSize = data.X.getRows();

  assert(O.size() == _dims.size());

  /*for (size_t i=0; i<_transforms.size(); ++i)
    _transforms[i].feedForward(O[i+1], O[i], offset, batchSize);*/

  O[0].resize(batchSize, data.X.getCols() + 1);

  memcpy2D(O[0], data.X, offset, 0, batchSize, data.X.getCols(), 0, 0);
  fillLastColumnWith(O[0], (float) 1.0);

  size_t end = O.size() - 1;
  for (size_t i=0; i<end - 1; ++i) {
    O[i+1] = ext::sigmoid(O[i] * _transforms[i].getW());
    fillLastColumnWith(O[i+1], (float) 1.0);
  }

  O[end] = ext::sigmoid(O[end - 1] * _transforms[end - 1].getW());
}

// ============================
// ===== Back Propagation =====
// ============================

void DNN::backPropagate(const DataSet& data, std::vector<mat>& O, size_t offset, size_t nData) {
  // mat error = O.back() - train.y;
  mat delta = calcError(O.back(), data.y, offset, nData);

  for (int i=_transforms.size() - 1; i >= 0; --i) {

    _transforms[i].getDw() = ~O[i] * delta;
    // delta *= ~_w[i];
    
    //   delta = delta(:, 1:end-1) * ~_w[i]
    //
    //                  (temp)
    //     delta'    =  delta    x     (weigth)^T
    // -------------------------------------------
    //       7                             7
    // |<--------->|   ----->|       |<--------->|
    // o o o o o o o = o o o o o x | o o o o o o o 
    // o o o o o o o   o o o o o   | o o o o o o o 
    // o o o o o o o   o o o o o   | o o o o o o o 
    //                             v o o o o o o o 
    //                               o o o o o o o  (<== bias, don't use them when back-propagate)

    size_t D1 = _transforms[i].getW().getRows() - 1,
           D2 = (i == _transforms.size() - 1) ? delta.getCols() 
					   : delta.getCols() - 1,
           nData = delta.getRows();

    mat tmp(delta);
    delta.resize(nData, D1 + 1);

    device_matrix<float>::cublas_gemm(
	HIPBLAS_OP_N, HIPBLAS_OP_T,
	nData, D1 + 1, D2 /* Ignore last column, which is the bias */,
	1.0,
	tmp.getData(), nData,
	_transforms[i].getW().getData(), D1 + 1,
	0.0,
	delta.getData(), nData);
    
    thrust::device_vector<float> temp(O[i].size());

    thrust::device_ptr<float> output(O[i].getData());
    thrust::transform(output, output + O[i].size(), temp.begin(), func::dsigma<float>());

    thrust::device_ptr<float> dv1(delta.getData());
    thrust::transform(dv1, dv1 + delta.size(), temp.begin(), dv1, thrust::multiplies<float>());
  }
}

void DNN::updateParameters(float learning_rate) { 
  for (size_t i=0; i<_transforms.size(); ++i)
    _transforms[i].update(learning_rate);
}

void swap(DNN& lhs, DNN& rhs) {
  using WHERE::swap;
  swap(lhs._dims, rhs._dims);
  swap(lhs._transforms, rhs._transforms);
}

// =============================
// ===== Utility Functions =====
// =============================

mat l2error(mat& targets, mat& predicts) {
  mat err(targets - predicts);

  thrust::device_ptr<float> ptr(err.getData());
  thrust::transform(ptr, ptr + err.size(), ptr, func::square<float>());

  mat sum_matrix(err.getCols(), 1);
  err *= sum_matrix;
  
  return err;
}


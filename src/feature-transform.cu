#include "hip/hip_runtime.h"
#include <feature-transform.h>

AffineTransform::AffineTransform(): _isOutputLayer(false) {}

AffineTransform::AffineTransform(const AffineTransform& source):
  _isOutputLayer(source._isOutputLayer),
  _w(source._w),
  _dw(source._dw) {

  }

AffineTransform::AffineTransform(const mat& w): _w(w), _dw(w.getRows(), w.getCols()), _isOutputLayer(false) {

}

AffineTransform::AffineTransform(size_t rows, size_t cols): _w(rows, cols), _dw(rows, cols), _isOutputLayer(false) {
  ext::randn(_w);
}

AffineTransform& AffineTransform::operator = (AffineTransform rhs) {
  swap(*this, rhs);
  return *this;
}

void AffineTransform::setOutputLayer(bool flag) {
  _isOutputLayer = flag;
}

mat& AffineTransform::getW() {
  return _w;
}

const mat& AffineTransform::getW() const {
  return _w;
}

mat& AffineTransform::getDw() {
  return _dw;
}

const mat& AffineTransform::getDw() const {
  return _dw;
}

void AffineTransform::update(float learning_rate) {
  _dw *= learning_rate;
  _w -= _dw;
}

void AffineTransform::resize(size_t rows, size_t cols) {
  _w.resize(rows, cols);
  _dw.resize(rows, cols);
}

string AffineTransform::toString() const {
  return "affinetransform";
}

void AffineTransform::feedForward(mat& fout, const mat& fin, size_t offset, size_t nData) {
  fout = ext::sigmoid(const_cast<mat&>(fin) * _w);
  fillLastColumnWith(fout, (float) 1.0);
}

void AffineTransform::backPropagate(const mat& fin, const mat& fout, mat& error) {

  mat delta = error & (1 - fout) & fout;

  _dw = ~const_cast<mat&>(fin) * delta;

  // Ignore last column, which is the bias
  size_t traceLength = delta.getCols() - 1;

  error.resize(delta.getRows(), _w.getRows());

  device_matrix<float>::cublas_gemm(
      HIPBLAS_OP_N, HIPBLAS_OP_T,
      delta.getRows(), _w.getRows(), traceLength, 
      1.0,
      delta.getData(), delta.getRows(),
      _w.getData(), _w.getRows(),
      0.0,
      error.getData(), error.getRows());
}

void swap(AffineTransform& lhs, AffineTransform& rhs) {
  std::swap(lhs._isOutputLayer, rhs._isOutputLayer);
  std::swap(lhs._w, rhs._w);
  std::swap(lhs._dw, rhs._dw);
}

// ===================
// ===== Softmax =====
// ===================

Softmax::Softmax(const mat& w): AffineTransform(w) {
}

Softmax::Softmax(size_t rows, size_t cols): AffineTransform(rows, cols) {
}

Softmax& Softmax::operator = (Softmax rhs) {
  AffineTransform::operator=(rhs);
  swap(*this, rhs);
  return *this;
}

string Softmax::toString() const {
  return "softmax";
}

__global__ void substract_max_per_row(float* const A, unsigned int rows, unsigned int cols) {
  extern __shared__ float sdata[];

  // Matrix index
  int ty = threadIdx.y;
  int x = threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;

  if (x >= cols || y >= rows)
    return;

  sdata[x * blockDim.y + ty] = A[x * rows + y];

  for (unsigned int s = blockDim.x/2 ; s > 0; s >>= 1) {
    if (x < s && x + s < cols) {
      if (sdata[(x + s) * blockDim.y + ty] > sdata[x * blockDim.y + ty])
	sdata[x * blockDim.y + ty] = sdata[(x + s) * blockDim.y + ty];
    }
    __syncthreads();
  }

  A[x * rows + y] -= sdata[ty];
}

void substractMaxPerRow(mat& x) {
  size_t rows = x.getRows(),
	 cols = x.getCols();

  const size_t N = 32;
  assert(cols <= N);

  dim3 grid;
  grid.x = 1;
  grid.y = (unsigned int) ceil((float) rows / N);
  dim3 threads(N, N);

  size_t smSize = N * N * sizeof(float);

  substract_max_per_row<<< grid, threads, smSize >>>(x.getData(), rows, cols);
  CCE(hipDeviceSynchronize());
}


void Softmax::feedForward(mat& fout, const mat& fin, size_t offset, size_t nData) {

  mat x = const_cast<mat&>(fin) * const_cast<mat&>(_w);
  x.resize(x.getRows(), x.getCols() - 1);
  substractMaxPerRow(x);

  mat p(x.getRows(), x.getCols());

  thrust::device_ptr<float> xPtr(x.getData());
  thrust::device_ptr<float> pPtr(p.getData());
  thrust::transform(xPtr, xPtr + x.size(), pPtr, func::exp<float>());

  mat sumOfProb = p * (mat(p.getCols(), p.getCols()) += 1);

  fout.resize(p.getRows(), p.getCols() + 1);
  thrust::device_ptr<float> foutPtr(fout.getData());
  thrust::device_ptr<float> sPtr(sumOfProb.getData());
  thrust::transform(pPtr, pPtr + p.size(), sPtr, foutPtr, thrust::divides<float>());

  /*
  mat x = const_cast<mat&>(fin) * _w;
  x.resize(x.getRows(), x.getCols() - 1);

  std::vector<float> hx = copyToHost(x);

  float* h_max = new float[x.getRows()];

  for (size_t i=0; i<x.getRows(); ++i) {
    float m = hx[i];
    for (size_t j=0; j<x.getCols(); ++j) {
      float v = hx[j * x.getRows() + i];
      if (v > m)
	m = v;
    }
    h_max[i] = m;
  }

  mat d_max = mat(h_max, x.getRows(), 1) * (mat(1, x.getCols()) += 1);
  delete [] h_max;
  x -= d_max;

  mat p(x.getRows(), x.getCols());

  thrust::device_ptr<float> xPtr(x.getData());
  thrust::device_ptr<float> pPtr(p.getData());
  thrust::transform(xPtr, xPtr + x.size(), pPtr, func::exp<float>());

  mat sumOfProb = p * (mat(p.getCols(), p.getCols()) += 1);

  fout.resize(p.getRows(), p.getCols() + 1);
  thrust::device_ptr<float> foutPtr(fout.getData());
  thrust::device_ptr<float> sPtr(sumOfProb.getData());
  thrust::transform(pPtr, pPtr + p.size(), sPtr, foutPtr, thrust::divides<float>());
  */
}

mat rowSum(mat& m) {
  return m * (mat(m.getCols(), m.getCols()) += 1);
}

void Softmax::backPropagate(const mat& fin, const mat& fout, mat& error) {

  mat error_times_fout = error & fout;
  mat sum = rowSum(error_times_fout);

  mat sum_times_fout = sum & fout;
  mat delta = error_times_fout - sum_times_fout;

  _dw = ~const_cast<mat&>(fin) * delta;

  // Ignore last column, which is the bias
  size_t traceLength = delta.getCols() - 1;

  error.resize(delta.getRows(), _w.getRows());

  device_matrix<float>::cublas_gemm(
      HIPBLAS_OP_N, HIPBLAS_OP_T,
      delta.getRows(), _w.getRows(), traceLength, 
      1.0,
      delta.getData(), delta.getRows(),
      _w.getData(), _w.getRows(),
      0.0,
      error.getData(), error.getRows());
}

void swap(Softmax& lhs, Softmax& rhs) {
}


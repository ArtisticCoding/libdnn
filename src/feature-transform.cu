#include "hip/hip_runtime.h"
#include <feature-transform.h>

void substractMaxPerRow(mat& x);

// ============================
// ===== FeatureTransform =====
// ============================

FeatureTransform::FeatureTransform(const FeatureTransform& source): _w(source._w), _dw(source._dw) {
}

FeatureTransform::FeatureTransform(const mat& w): _w(w), _dw(w.getRows(), w.getCols()) {
}

FeatureTransform::FeatureTransform(size_t rows, size_t cols, float variance): _w(rows, cols), _dw(rows, cols) {
  ext::randn(_w, 0.0f, variance);
}

mat& FeatureTransform::getW() {
  return _w;
}

const mat& FeatureTransform::getW() const {
  return _w;
}

mat& FeatureTransform::getDw() {
  return _dw;
}

const mat& FeatureTransform::getDw() const {
  return _dw;
}

void FeatureTransform::update(float learning_rate) {
  _dw *= learning_rate;
  _w -= _dw;
}

// ===================
// ===== Sigmoid =====
// ===================

Sigmoid::Sigmoid(const mat& w): FeatureTransform(w) {
}

Sigmoid::Sigmoid(size_t rows, size_t cols, float variance): FeatureTransform(rows, cols, variance) {
}

Sigmoid::Sigmoid(const Sigmoid& src): FeatureTransform(src) {
}

Sigmoid* Sigmoid::clone() const {
  return new Sigmoid(*this);
}

string Sigmoid::toString() const {
  return "sigmoid";
}

void Sigmoid::feedForward(mat& fout, const mat& fin, size_t offset, size_t nData) {
  fout = ext::sigmoid(const_cast<mat&>(fin) * _w);
  fillLastColumnWith(fout, (float) 1.0);
}

void Sigmoid::backPropagate(const mat& fin, const mat& fout, mat& error) {
  mat delta = error & (1 - fout) & fout;

  _dw = ~const_cast<mat&>(fin) * delta;

  // Ignore last column, which is the bias
  size_t traceLength = delta.getCols() - 1;

  error.resize(delta.getRows(), _w.getRows());

  device_matrix<float>::cublas_gemm(
      HIPBLAS_OP_N, HIPBLAS_OP_T,
      delta.getRows(), _w.getRows(), traceLength, 
      1.0,
      delta.getData(), delta.getRows(),
      _w.getData(), _w.getRows(),
      0.0,
      error.getData(), error.getRows());
}

// ===================
// ===== Softmax =====
// ===================

Softmax::Softmax(const mat& w): FeatureTransform(w) {
}

Softmax::Softmax(size_t rows, size_t cols, float variance): FeatureTransform(rows, cols, variance) {
}

Softmax::Softmax(const Softmax& src): FeatureTransform(src) {
}

Softmax* Softmax::clone() const {
  return new Softmax(*this);
}

string Softmax::toString() const {
  return "softmax";
}

__global__ void substract_max_per_row(float* const A, unsigned int rows, unsigned int cols) {
  extern __shared__ float sdata[];

  // Matrix index
  int ty = threadIdx.y;
  int x = threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;

  if (x >= cols || y >= rows)
    return;

  sdata[x * blockDim.y + ty] = A[x * rows + y];

  for (unsigned int s = blockDim.x/2 ; s > 0; s >>= 1) {
    if (x >= s || x + s >= cols)
      continue;

    if (sdata[(x + s) * blockDim.y + ty] > sdata[x * blockDim.y + ty])
      sdata[x * blockDim.y + ty] = sdata[(x + s) * blockDim.y + ty];

    __syncthreads();
  }

  A[x * rows + y] -= sdata[ty];
}

void substractMaxPerRow(mat& x) {
  size_t rows = x.getRows(),
	 cols = x.getCols();

  const size_t N = 32;
  assert(cols <= N);

  dim3 grid;
  grid.x = 1;
  grid.y = (unsigned int) ceil((float) rows / N);
  dim3 threads(N, N);

  size_t smSize = N * N * sizeof(float);

  substract_max_per_row<<< grid, threads, smSize >>>(x.getData(), rows, cols);
  CCE(hipDeviceSynchronize());
}

void Softmax::feedForward(mat& fout, const mat& fin, size_t offset, size_t nData) {

  mat x = const_cast<mat&>(fin) * const_cast<mat&>(_w);
  x.resize(x.getRows(), x.getCols() - 1);
  substractMaxPerRow(x);

  mat p(x.getRows(), x.getCols());

  thrust::device_ptr<float> xPtr(x.getData());
  thrust::device_ptr<float> pPtr(p.getData());
  thrust::transform(xPtr, xPtr + x.size(), pPtr, func::exp<float>());

  mat sumOfProb = p * (mat(p.getCols(), p.getCols()) += 1);

  fout.resize(p.getRows(), p.getCols() + 1);
  thrust::device_ptr<float> foutPtr(fout.getData());
  thrust::device_ptr<float> sPtr(sumOfProb.getData());
  thrust::transform(pPtr, pPtr + p.size(), sPtr, foutPtr, thrust::divides<float>());
}

mat rowSum(mat& m) {
  return m * (mat(m.getCols(), m.getCols()) += 1);
}

void Softmax::backPropagate(const mat& fin, const mat& fout, mat& error) {

  mat error_times_fout = error & fout;
  mat sum = rowSum(error_times_fout);

  mat sum_times_fout = sum & fout;
  mat delta = error_times_fout - sum_times_fout;

  _dw = ~const_cast<mat&>(fin) * delta;

  // Ignore last column, which is the bias
  size_t traceLength = delta.getCols() - 1;

  error.resize(delta.getRows(), _w.getRows());

  device_matrix<float>::cublas_gemm(
      HIPBLAS_OP_N, HIPBLAS_OP_T,
      delta.getRows(), _w.getRows(), traceLength, 
      1.0,
      delta.getData(), delta.getRows(),
      _w.getData(), _w.getRows(),
      0.0,
      error.getData(), error.getRows());
}

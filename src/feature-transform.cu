#include "hip/hip_runtime.h"
#include <feature-transform.h>

void FeatureTransform::print(FILE* fid, const host_matrix<float>& data, string type) {
  fprintf(fid, "<%s> %lu %lu\n", type.c_str(), data.getRows() - 1, data.getCols() - 1);

  size_t rows = data.getRows(),
	 cols = data.getCols();

  fprintf(fid, " [");

  for (size_t j=0; j<rows-1; ++j) {
    fprintf(fid, "\n  ");
    for (size_t k=0; k<cols-1; ++k)
      fprintf(fid, "%g ", data[k * rows + j]);
  }
  fprintf(fid, "]\n");

  fprintf(fid, "<bias> \n [");
  for (size_t j=0; j<cols-1; ++j)
    fprintf(fid, "%g ", data[j * rows + rows - 1]);
  fprintf(fid, " ]\n");
}

mat rowSum(mat& m) {
  return m * mat(m.getCols(), m.getCols(), 1);
}

// convert a linear index to a row index
template <typename T>
struct linear_index_to_row_index : public thrust::unary_function<T,T>
{
  T C; // number of columns

  __host__ __device__
    linear_index_to_row_index(T C) : C(C) {}

  __host__ __device__
    T operator()(T i)
    {
      return i / C;
    }
};

void substractMaxPerRow(mat& x);
mat getRowMax(mat& A);
__global__ void substract_max_per_row(float* const A, float* const rmax, unsigned int rows, unsigned int cols);

void substractMaxPerRow(mat& x) {
  mat rmax = getRowMax(x);

  ALLOCATE_GRIDS_AND_THREADS(x.getRows(), x.getCols());
  substract_max_per_row<<< grids, threads >>>(x.getData(), rmax.getData(), x.getRows(), x.getCols());
  CCE(hipDeviceSynchronize());
}

__global__ void substract_max_per_row(float* const A, float* const rmax, unsigned int rows, unsigned int cols) {
  // Matrix index
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;

  if (x >= cols || y >= rows)
    return;

  A[x * rows + y] -= rmax[y];
}

mat getRowMax(mat& A) {
  mat rmax(A.getRows(), 1);
  mat At = ~A;

  // allocate storage for per-row results and indices
  thrust::device_vector< float > row_indices(A.getRows());
  thrust::device_vector< float > row_results(A.getRows());

  // compute row sums by summing values with equal row indices
  thrust::reduce_by_key
    (thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(A.getCols())),
     thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(A.getCols())) + A.size(),
     thrust::device_ptr<float>(At.getData()),
     row_indices.begin(),
     thrust::device_ptr<float>(rmax.getData()),
     thrust::equal_to<float>(),
     thrust::maximum<float>());

  return rmax;
}

// ============================
// ===== FeatureTransform =====
// ============================

FeatureTransform::FeatureTransform(const FeatureTransform& source): _w(source._w) {
}

FeatureTransform::FeatureTransform(const mat& w): _w(w){
}

size_t FeatureTransform::getInputDimension() const {
  return _w.getRows();
}

size_t FeatureTransform::getOutputDimension() const {
  return _w.getCols();
}

void FeatureTransform::print(FILE* fid) const {
  FeatureTransform::print(fid, this->_w, this->toString());
}

void FeatureTransform::feedBackward(mat& error, const mat& delta) {
  // The last row of _w is bias, and the last column of _w is saved only for computational efficiency.
  // Therefore, ignore last column, which is the bias.
  size_t traceLength = delta.getCols() - 1;

  error.resize(delta.getRows(), _w.getRows());

  device_matrix<float>::cublas_gemm(
      HIPBLAS_OP_N, HIPBLAS_OP_T,
      delta.getRows(), _w.getRows(), traceLength, 
      1.0,
      delta.getData(), delta.getRows(),
      _w.getData(), _w.getRows(),
      0.0,
      error.getData(), error.getRows());
}

// ===================
// ===== Sigmoid =====
// ===================

Sigmoid::Sigmoid(const mat& w): FeatureTransform(w) {
}

Sigmoid::Sigmoid(const Sigmoid& src): FeatureTransform(src) {
}

Sigmoid* Sigmoid::clone() const {
  return new Sigmoid(*this);
}

string Sigmoid::toString() const {
  return "sigmoid";
}

void Sigmoid::feedForward(mat& fout, const mat& fin) {
  // fout = sigmoid(fin * _w);
  fout = transform(fin * _w, func::sigmoid<float>());
  fillLastColumnWith(fout, (float) 1.0);
}

void Sigmoid::backPropagate(mat& error, const mat& fin, const mat& fout, float learning_rate) {
  mat delta = error & (1.0f - fout) & fout;
  this->feedBackward(error, delta);
  gemm(fin, delta, _w, -learning_rate, 1.0f, true, false);
}

// ===================
// ===== Softmax =====
// ===================

Softmax::Softmax(const mat& w): FeatureTransform(w) {
}

Softmax::Softmax(const Softmax& src): FeatureTransform(src) {
}

Softmax* Softmax::clone() const {
  return new Softmax(*this);
}

string Softmax::toString() const {
  return "softmax";
}

void Softmax::feedForward(mat& fout, const mat& fin) {

  mat x = fin * _w;
  x.resize(x.getRows(), x.getCols() - 1);
  substractMaxPerRow(x);

  mat p(x.getRows(), x.getCols());

  thrust::device_ptr<float> xPtr(x.getData());
  thrust::device_ptr<float> pPtr(p.getData());
  thrust::transform(xPtr, xPtr + x.size(), pPtr, func::exp<float>());

  mat sumOfProb = p * mat(p.getCols(), p.getCols(), 1);

  fout.resize(p.getRows(), p.getCols() + 1);
  thrust::device_ptr<float> foutPtr(fout.getData());
  thrust::device_ptr<float> sPtr(sumOfProb.getData());
  thrust::transform(pPtr, pPtr + p.size(), sPtr, foutPtr, thrust::divides<float>());
}

void Softmax::backPropagate(mat& error, const mat& fin, const mat& fout, float learning_rate) {

  // This is much faster and easier
  mat delta = error;
  this->feedBackward(error, delta);
  gemm(fin, delta, _w, -learning_rate, 1.0f, true, false);

  // cf. /usr/local/lib/python2.7/dist-packages/theano/tensor/nnet/nnet.py:251
  /*mat error_times_fout = error & fout;
  mat delta = error_times_fout - (rowSum(error_times_fout) & fout);

  this->feedBackward(error, delta);

  gemm(fin, delta, _w, -learning_rate, 1.0f, true, false);*/
}

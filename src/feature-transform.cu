#include "hip/hip_runtime.h"
#include <feature-transform.h>

// convert a linear index to a row index
template <typename T>
struct linear_index_to_row_index : public thrust::unary_function<T,T>
{
  T C; // number of columns

  __host__ __device__
    linear_index_to_row_index(T C) : C(C) {}

  __host__ __device__
    T operator()(T i)
    {
      return i / C;
    }
};

void substractMaxPerRow(mat& x);
mat getRowMax(mat& A);
__global__ void substract_max_per_row(float* const A, float* const rmax, unsigned int rows, unsigned int cols);

void substractMaxPerRow(mat& x) {
  mat rmax = getRowMax(x);

  const int N = 32;
  dim3 grid;
  grid.x = (unsigned int) ceil((float) x.getCols() / N);
  grid.y = (unsigned int) ceil((float) x.getRows() / N);
  dim3 threads(N, N);

  substract_max_per_row<<<grid, threads>>>(x.getData(), rmax.getData(), x.getRows(), x.getCols());
  CCE(hipDeviceSynchronize());
}

__global__ void substract_max_per_row(float* const A, float* const rmax, unsigned int rows, unsigned int cols) {
  // Matrix index
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;

  if (x >= cols || y >= rows)
    return;

  A[x * rows + y] -= rmax[y];
}

mat getRowMax(mat& A) {
  mat rmax(A.getRows(), 1);
  mat At = ~A;

  // allocate storage for per-row results and indices
  thrust::device_vector< float > row_indices(A.getRows());
  thrust::device_vector< float > row_results(A.getRows());

  // compute row sums by summing values with equal row indices
  thrust::reduce_by_key
    (thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(A.getCols())),
     thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(A.getCols())) + A.size(),
     thrust::device_ptr<float>(At.getData()),
     row_indices.begin(),
     thrust::device_ptr<float>(rmax.getData()),
     thrust::equal_to<float>(),
     thrust::maximum<float>());

  return rmax;
}

string toString(std::vector<float> data, size_t rows, size_t cols) {
  stringstream ss;
  ss << " [";

  for (size_t j=0; j<rows-1; ++j) {
    ss << "\n  ";
    for (size_t k=0; k<cols; ++k)
      ss << data[k * rows + j] << " ";
  }
  ss << "]\n";

  ss << "<bias> \n [";
  for (size_t j=0; j<cols; ++j)
    ss << data[j * rows + rows - 1] << " ";
  ss << " ]\n";

  return ss.str();
}

// ============================
// ===== FeatureTransform =====
// ============================

FeatureTransform::FeatureTransform(const FeatureTransform& source): _w(source._w), _dw(source._dw) {
}

FeatureTransform::FeatureTransform(const mat& w): _w(w), _dw(w.getRows(), w.getCols()) {
}

FeatureTransform::FeatureTransform(size_t rows, size_t cols, float variance): _w(rows, cols), _dw(rows, cols) {
  ext::randn(_w, 0.0f, variance);
}

mat& FeatureTransform::getW() {
  return _w;
}

const mat& FeatureTransform::getW() const {
  return _w;
}

mat& FeatureTransform::getDw() {
  return _dw;
}

const mat& FeatureTransform::getDw() const {
  return _dw;
}

void FeatureTransform::update(float learning_rate) {
  _dw *= learning_rate;
  _w -= _dw;
}

// ===================
// ===== Sigmoid =====
// ===================

Sigmoid::Sigmoid(const mat& w): FeatureTransform(w) {
}

Sigmoid::Sigmoid(size_t rows, size_t cols, float variance): FeatureTransform(rows, cols, variance) {
}

Sigmoid::Sigmoid(const Sigmoid& src): FeatureTransform(src) {
}

Sigmoid* Sigmoid::clone() const {
  return new Sigmoid(*this);
}

string Sigmoid::toString() const {
  size_t rows = _w.getRows(),
	 cols = _w.getCols() - 1;

  stringstream ss;
  ss << "<sigmoid> " << rows - 1 << " " << cols << endl;
  ss << ::toString(copyToHost(_w), rows, cols);
  return ss.str();
}

void Sigmoid::feedForward(mat& fout, const mat& fin) {
  fout = ext::sigmoid(const_cast<mat&>(fin) * _w);
  fillLastColumnWith(fout, (float) 1.0);
}

void Sigmoid::backPropagate(mat& error, const mat& fin, const mat& fout) {
  mat delta = error & (1.0f - fout) & fout;

  _dw = ~const_cast<mat&>(fin) * delta;

  // Ignore last column, which is the bias
  size_t traceLength = delta.getCols() - 1;

  error.resize(delta.getRows(), _w.getRows());

  device_matrix<float>::cublas_gemm(
      HIPBLAS_OP_N, HIPBLAS_OP_T,
      delta.getRows(), _w.getRows(), traceLength, 
      1.0,
      delta.getData(), delta.getRows(),
      _w.getData(), _w.getRows(),
      0.0,
      error.getData(), error.getRows());
}

// ===================
// ===== Softmax =====
// ===================

Softmax::Softmax(const mat& w): FeatureTransform(w) {
}

Softmax::Softmax(size_t rows, size_t cols, float variance): FeatureTransform(rows, cols, variance) {
}

Softmax::Softmax(const Softmax& src): FeatureTransform(src) {
}

Softmax* Softmax::clone() const {
  return new Softmax(*this);
}

string Softmax::toString() const {
  size_t rows = _w.getRows(),
	 cols = _w.getCols() - 1;

  stringstream ss;
  ss << "<softmax> " << rows - 1 << " " << cols << endl;
  ss << ::toString(copyToHost(_w), rows, cols);
  return ss.str();
}

/*__global__ void substract_max_per_row(float* const A, unsigned int rows, unsigned int cols) {
  extern __shared__ float sdata[];

  // Matrix index
  int ty = threadIdx.y;
  int tx = threadIdx.x;
  int x = threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;

  if (x >= cols || y >= rows)
    return;

  unsigned int idx = x * blockDim.y + ty;
  sdata[idx] = A[x * rows + y];

  for (unsigned int s = blockDim.x/2 ; s > 0; s >>= 1) {
    if (x >= s || x + s >= cols)
      continue;

    if (sdata[(x + s) * blockDim.y + ty] > sdata[idx])
      sdata[idx] = sdata[(x + s) * blockDim.y + ty];

    __syncthreads();
  }

  A[x * rows + y] -= sdata[ty];
}

void substractMaxPerRow(mat& x) {
  size_t rows = x.getRows(),
	 cols = x.getCols();

  const size_t N = 32;
  assert(cols <= N);

  dim3 grid;
  grid.x = 1;
  grid.y = (unsigned int) ceil((float) rows / N);
  dim3 threads(N, N);

  size_t smSize = N * N * sizeof(float);

  substract_max_per_row<<< grid, threads, smSize >>>(x.getData(), rows, cols);
  CCE(hipDeviceSynchronize());
}*/

void Softmax::feedForward(mat& fout, const mat& fin) {

  mat x = const_cast<mat&>(fin) * const_cast<mat&>(_w);
  x.resize(x.getRows(), x.getCols() - 1);
  substractMaxPerRow(x);

  mat p(x.getRows(), x.getCols());

  thrust::device_ptr<float> xPtr(x.getData());
  thrust::device_ptr<float> pPtr(p.getData());
  thrust::transform(xPtr, xPtr + x.size(), pPtr, func::exp<float>());

  mat sumOfProb = p * (mat(p.getCols(), p.getCols()) += 1);

  fout.resize(p.getRows(), p.getCols() + 1);
  thrust::device_ptr<float> foutPtr(fout.getData());
  thrust::device_ptr<float> sPtr(sumOfProb.getData());
  thrust::transform(pPtr, pPtr + p.size(), sPtr, foutPtr, thrust::divides<float>());
}

mat rowSum(mat& m) {
  return m * (mat(m.getCols(), m.getCols()) += 1);
}

void Softmax::backPropagate(mat& error, const mat& fin, const mat& fout) {

  mat error_times_fout = error & fout;
  mat sum = rowSum(error_times_fout);

  mat sum_times_fout = sum & fout;
  mat delta = error_times_fout - sum_times_fout;

  _dw = ~const_cast<mat&>(fin) * delta;

  // Ignore last column, which is the bias
  size_t traceLength = delta.getCols() - 1;

  error.resize(delta.getRows(), _w.getRows());

  device_matrix<float>::cublas_gemm(
      HIPBLAS_OP_N, HIPBLAS_OP_T,
      delta.getRows(), _w.getRows(), traceLength, 
      1.0,
      delta.getData(), delta.getRows(),
      _w.getData(), _w.getRows(),
      0.0,
      error.getData(), error.getRows());
}

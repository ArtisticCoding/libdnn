#include "hip/hip_runtime.h"
#include <rbm.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#define fill_bias(x) { fillLastColumnWith(x, 1.0f); }

void playground() {
  perf::Timer timer;
  timer.start();

  mat X(1024, 1024);
  for (int i=0; i<10000; ++i)
    addGaussian(X);

  timer.elapsed();

  // Prevent O3 optimization
  X.save("x.mat");
}

ostream& operator << (ostream& os, const RBM_TYPE& type) {
  switch (type) {
    case GAUSSIAN_BERNOULLI:
      os << "Gaussian-Bernoulli RBM"; break;
    case BERNOULLI_BERNOULLI:
      os << "Bernoulli-Bernoulli RBM"; break;
  }
  return os;
}

hmat batchFeedForwarding(const hmat& X, const mat& w) {
  size_t nData = X.getCols();

  hmat Y(w.getCols(), nData);
  Batches batches(2048, nData);
  for (Batches::iterator itr = batches.begin(); itr != batches.end(); ++itr) {
    mat fin  = getBatchData(X, *itr);
    mat fout = sigmoid(fin * w);
    fill_bias(fout);

    size_t offset = fout.getCols() * itr->offset,
	   nBytes = sizeof(float) * fout.size();

    fout = ~fout;
    CCE(hipMemcpy(Y.getData() + offset, fout.getData(), nBytes, hipMemcpyDeviceToHost));
  }
  return Y;
}

std::vector<mat> initStackedRBM(DataSet& data, const std::vector<size_t>& dims, float slopeThres, RBM_TYPE type) {
  std::vector<mat> weights(dims.size() - 1);

  size_t nData = data.size();

  hmat X = data.getX();
  for (size_t i=0; i<weights.size(); ++i) {
    // Only the first layer need to be Gaussian-Bernoulli
    if (type == GAUSSIAN_BERNOULLI && i > 0)
      type = BERNOULLI_BERNOULLI;

    weights[i] = rbmTrain(X, dims[i + 1], slopeThres, type);
    X = batchFeedForwarding(X, weights[i]);
  }

  return weights;
}

__device__ float generate_randn(hiprandState* globalState) {
  hiprandState localState = *globalState;
  float RANDOM = hiprand_normal( &localState );
  *globalState = localState;
  return RANDOM;
}

__device__ float generate_rand(hiprandState* globalState) {
  hiprandState localState = *globalState;
  float RANDOM = hiprand_uniform( &localState );
  *globalState = localState;
  return RANDOM;
}

__global__ void setupCuRandState( hiprandState * state, unsigned long seed ) {
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  hiprand_init ( seed, x, 0, &state[x] );
}

__global__ void add_gaussian_kernel(float* const data, hiprandState* globalState, unsigned int rows, unsigned int cols) {
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Matrix index
  int x = blockIdx.x*blockDim.x + tx;
  int y = blockIdx.y*blockDim.y + ty;

  if (x >= cols || y >= rows)
    return;

  int i = x * rows + y;
  int j = tx * blockDim.y + ty;
  data[i] = (float) (data[i] + generate_randn(globalState + j));
  // data[i] = (float) (data[i] + hiprand_uniform(globalState + j));
  __syncthreads();
}

__global__ void sample_kernel(float* const data, hiprandState* globalState, unsigned int rows, unsigned int cols) {
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Matrix index
  int x = blockIdx.x*blockDim.x + tx;
  int y = blockIdx.y*blockDim.y + ty;

  if (x >= cols || y >= rows)
    return;

  int i = x * rows + y;
  int j = tx * blockDim.y + ty;
  data[i] = (float) (data[i] > generate_rand(globalState + j));
  __syncthreads();
}

class CURAND_STATE {
public:
  CURAND_STATE(unsigned seed = unsigned(time(NULL)), int N = 32): _states(NULL) {
    hipMalloc ( &_states, N * N * sizeof( hiprandState ) );
    setupCuRandState <<< 1, N * N >>> ( _states, seed );
  }

  hiprandState* get() const { return _states; }

  ~CURAND_STATE() {
    hipFree(_states);
  }

private:
  hiprandState* _states;
};

void sample(mat &prob) {
  static CURAND_STATE state;

  const size_t N = 32;
  dim3 threads(N, N);
  dim3 grid;
  grid.x = (unsigned int) ceil((float) prob.getCols() / N);
  grid.y = (unsigned int) ceil((float) prob.getRows() / N);

  sample_kernel<<< grid, threads >>>(prob.getData(), state.get(), prob.getRows(), prob.getCols());
  CCE(hipDeviceSynchronize());
}

void addGaussian(mat &prob) {
  static CURAND_STATE state;

  const size_t N = 32;
  dim3 threads(N, N);
  dim3 grid;
  grid.x = (unsigned int) ceil((float) prob.getCols() / N);
  grid.y = (unsigned int) ceil((float) prob.getRows() / N);

  add_gaussian_kernel<<< grid, threads >>>(prob.getData(), state.get(), prob.getRows(), prob.getCols());
  CCE(hipDeviceSynchronize());
}

void apply_cmvn(hmat& data) {
  size_t input_dim = data.getRows();
  size_t nData = data.getCols();

  for (int i=0; i<input_dim - 1; ++i) {
    float mean = 0;
    for (int j=0; j<nData; ++j)
      mean += data(i, j);
    mean /= nData;

    for (int j=0; j<nData; ++j)
      data(i, j) -= mean;

    if (nData <= 1)
      continue;

    float deviation = 0;
    for (int j=0; j<nData; ++j)
      deviation += pow(data(i, j), 2.0f);
    deviation = sqrt(deviation / (nData - 1));

    if (deviation == 0)
      continue;

    for (int j=0; j<nData; ++j)
      data(i, j) /= deviation;
  }
}

mat rbmTrain(const hmat& d, size_t nHiddenUnits, float threshold, RBM_TYPE type) {
  hmat data(d);

  float learningRate = 1e-1;
  switch (type) {
    case BERNOULLI_BERNOULLI:
      cout << "BERNOULLI_BERNOULLI" << endl;

      // If Bernoulli, make sure the visible units have values in the range [0, 1]
      assert(ext::max(d) <= 1 && ext::min(d) >= 0);
      break;
    case GAUSSIAN_BERNOULLI:
      cout << "GAUSSIAN_BERNOULLI" << endl;

      // Note: The learning rate of Gaussian RBM needs to be about one or two
      // orders of magnitude smaller than when using binary visible units.
      // Otherwise value will explode very quickly and get NaN.
      // [cf. A Practical Guide to Training Restricted Boltzmann Machines]
      learningRate /= 100;
      apply_cmvn(data);
      break;
  }

  size_t batchSize = 128;
  size_t input_dim = data.getRows();
  size_t nData = data.getCols();

  mat W(input_dim, nHiddenUnits + 1);
  ext::randn(W, 0, 0.1 / W.getCols());

  size_t minEpoch = 5, maxEpoch = 1024;

  std::vector<float> errors;
  errors.reserve(maxEpoch);

  float initialSlope = 0;

  ProgressBar pBar("RBM init ( error = ...       , slope ratio = ...        )");

  perf::Timer timer;
  timer.start();
  size_t epoch;
  for (epoch=0; epoch < maxEpoch; ++epoch) {

    float error = 0;

    Batches batches(batchSize, nData);
    for (Batches::iterator itr = batches.begin(); itr != batches.end(); ++itr) {

      mat v1, v2, h1, h2;
      v1 = getBatchData(data, *itr);
      fill_bias(v1);

      switch (type) {
	case BERNOULLI_BERNOULLI:
	  // Up Sampling
	  h1 = sigmoid(v1 * W);
	  sample(h1);
	  fill_bias(h1);

	  // Down-and-Up propagation
	  v2 = sigmoid(h1 * ~W);
	  fill_bias(v2);

	  h2 = sigmoid(v2 * W);
	  fill_bias(h2);

	  break;
	case GAUSSIAN_BERNOULLI:
	  // Up Sampling
	  h1 = v1 * W;
	  addGaussian(h1);
	  matlog(h1);
	  fill_bias(h1);

	  // Down-and-Up propagation
	  v2 = sigmoid(h1 * ~W);
	  fill_bias(v2);

	  h2 = v2 * W;
	  addGaussian(h2);
	  fill_bias(h2);

	  break;
      }

      // Calculate Positive & Negative
      mat positive = ~v1 * h1;
      mat negative = ~v2 * h2;

      float lr = learningRate / batchSize;

      mat dW = lr * (positive - negative);

      W += dW;
      error += pow(nrm2(v1 - v2), 2.0f);
    }

    errors.push_back(sqrt(error) / nData );

    if (epoch == minEpoch)
      initialSlope = getSlope(errors, minEpoch);

    if (epoch > minEpoch) {
      float ratio = abs(getSlope(errors, 5) / initialSlope);
      char status[100];
      sprintf(status, "RBM init ( error = %.4e, slope ratio = %.4e )", errors[epoch], ratio);
      pBar.refresh(std::min(1.0f, threshold / ratio), status);

      if (ratio < threshold)
	break;
    }
  }

  printf("Average magnitude of element in weight W = %.7f\n", nrm2(W) / sqrt(W.size()));
  float t_end = timer.getTime();
  printf("Average time for each epoch = %f\n", t_end / epoch);
  
  return W;
}

std::vector<size_t> getDimensionsForRBM(const DataSet& data, const string& structure) {

  string userInput = "";

  while (!is_number(userInput)) {
    printf("\33[33m Since RBM is a kind of UNSUPERVISED pre-training. "
	   "Please enter how many nodes you want in the output layer.\33[0m "
	   "[      ]\b\b\b\b\b");
    cin >> userInput;
  }

  size_t output_dim = atoi(userInput.c_str());

  // ===========================================================================
  // Initialize hidden structure
  size_t input_dim = data.getInputDimension();
  std::vector<size_t> dims = splitAsInt(structure, '-');
  dims.insert(dims.begin(), input_dim);
  dims.push_back((size_t) output_dim);

  printf("\n");
  printf("\33[32m Start RBM pre-training with following hidden structure:\33[0m\n");
  printf("\33[34m [   input  layer  ]\33[0m %lu\n", dims[0]);
  for (size_t i=1; i<dims.size()-1; ++i)
    printf("\33[34m [ hidden layer #%-2lu]\33[0m %lu\n", i, dims[i]);
  printf("\33[34m [   output layer  ]\33[0m %lu\n\n", dims.back());
  // ===========================================================================

  return dims;
}

float getSlope(const std::vector<float> &seq, size_t N) {
  std::vector<float> x(N);
  for (size_t i=0; i<N; ++i)
    x[i] = N - 1 - i;

  std::vector<float> y(N);
  for (size_t i=seq.size() - N; i<seq.size(); ++i)
    y[i - (seq.size() - N)] = seq[i];

  float m, c;
  linearRegression(x, y, &m, &c);

  return m;
}

float getAsymptoticBound(const std::vector<float> &error, size_t epoch, size_t maxEpoch, size_t N) {
  std::vector<float> x(N);
  for (size_t i=0; i<N; ++i)
    x[i] = epoch - (N - 1 - i);

  std::vector<float> y(N);
  for (size_t i=error.size() - N; i<error.size(); ++i)
    y[i - (error.size() - N)] = error[i];

  float m, c;
  linearRegression(x, y, &m, &c);

  return m * (float) maxEpoch + c;
}

/*mat sum(mat& m, size_t dimension = 1) {
  if (dimension == 1)
    return (mat(1, m.getRows()) += 1) * m;
  else
    return m * (mat(m.getCols(), 1) += 1);
} */

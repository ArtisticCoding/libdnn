#include "hip/hip_runtime.h"
#include <cnn-utility.h>

/*! Convert each row to a 2D image
 * \param data Each row in data is a feature vector. The number of rows in data
 *  is the number of feature vectors. The number of columns in data is the
 *  dimension of the feature vector.
 */
vector<mat> reshapeVectors2Images(const mat& data, const SIZE s) {

  mat t_data = ~data;
  vector<mat> images(data.getRows());

  for (size_t i=0; i<images.size(); ++i) {
    images[i].resize(s.m, s.n);

    CCE(hipMemcpy(images[i].getData(), t_data.getData() + i * t_data.getRows(),
	  sizeof(float) * images[i].size(), hipMemcpyDeviceToDevice));
  }

  return images;
}

mat reshapeImages2Vectors(const vector<mat>& images) {
  assert(images.size() > 0);

  mat t_data(images[0].size(), images.size());

  int M = images[0].getRows(),
      N = images[0].getCols();
  
  for (size_t i=0; i<images.size(); ++i)
    CCE(hipMemcpy(t_data.getData() + i * t_data.getRows(), images[i].getData(),
	  sizeof(float) * images[i].size(), hipMemcpyDeviceToDevice));

  return ~t_data;
}


SIZE parseInputDimension(const string &m_by_n) {
  size_t pos = m_by_n.find("x");
  return SIZE(str2int(m_by_n.substr(0, pos)), str2int(m_by_n.substr(pos+1)));
}

__global__ void convn_valid_kernel(float *output, float *data, float *kernel, int H, int W, int kH, int kW) { 
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Matrix index
  int x = blockIdx.x*blockDim.x + tx;
  int y = blockIdx.y*blockDim.y + ty;

  // vH, vW stands for valid H and valid W
  const int vH = H - kH + 1,
	    vW = W - kW + 1;

  if (x >= vW || y >= vH)
    return;

  x += kW - 1;
  y += kH - 1;

  float sum = 0; 
  for (int i = 0; i < kW; ++i)
    for(int j = 0; j < kH; ++j)
      sum += kernel[ i * kH + j ] * data[ (x - i) * H + (y - j) ]; 

  x -= kW - 1;
  y -= kH - 1;

  output[ x * vH + y ] = sum;
} 

__global__ void convn_same_kernel(float *output, float *data, float *kernel, int H, int W, int kH, int kW) { 
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Matrix index
  int x = blockIdx.x*blockDim.x + tx;
  int y = blockIdx.y*blockDim.y + ty;

  if (x >= W || y >= H)
    return;

  const int i0 = kW / 2, j0 = kH / 2;

  float sum = 0; 
  for (int i = 0; i < kW; ++i) {
    for(int j = 0; j < kH; ++j) {
      int ii = x - i + i0, jj = y - j + j0;

      if ( ii < 0 || ii >= W || jj < 0 || jj >= H )
	continue;

      sum += kernel[ i * kH + j ] * data[ ii * H + jj ]; 
    }
  }

  output[x * H + y] = sum;
} 


__global__ void convn_full_kernel(float *output, float *data, float *kernel, int H, int W, int kH, int kW) { 
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Matrix index
  int x = blockIdx.x*blockDim.x + tx;
  int y = blockIdx.y*blockDim.y + ty;

  // fH, fW stands for full H and full W
  const int fH = H + kH - 1,
	    fW = W + kW - 1;

  if (x >= fW || y >= fH)
    return;

  float sum = 0; 
  for (int i = 0; i < kW; ++i) {
    for(int j = 0; j < kH; ++j) {
      int ii = x - i, jj = y - j;

      if ( ii < 0 || ii >= W || jj < 0 || jj >= H )
	continue;

      sum += kernel[ i * kH + j ] * data[ ii * H + jj ]; 
    }
  }

  output[ x * fH + y ] = sum;
}

SIZE get_convn_size(const mat& data, const mat& kernel, string type) {

  int H = data.getRows(),
      W = data.getCols(),
      kH = kernel.getRows(),
      kW = kernel.getCols();

  if (type == "same")
    return SIZE(H, W);
  else if (type == "valid")
    return SIZE(max(H - kH + 1, 0), max(W - kW + 1, 0));
  else if (type == "full")
    return SIZE(H + kH - 1, W + kW - 1);
  else
    throw std::runtime_error("No such type of convolution");
}

mat convn(const mat& data, const mat& kernel, string type) {

  const int N_STREAM = 4;
  static vector<hipStream_t> streams(N_STREAM);
  static bool first = true;
  static int counter = 0;

  if (first) {
    first = false;
    for (size_t i=0; i<streams.size(); ++i)
      hipStreamCreate ( &streams[i] );
  }

  int H = data.getRows(),
      W = data.getCols(),
      kH = kernel.getRows(),
      kW = kernel.getCols();

  SIZE s = get_convn_size(data, kernel, type);
  mat output(s.m, s.n);

  ALLOCATE_GRIDS_AND_THREADS(output.getRows(), output.getCols());

  hipStream_t& stream = streams[counter];
  counter = (counter + 1) % N_STREAM;

  if (type == "same") {
    convn_same_kernel<<< grids, threads, 0, stream >>>(
	output.getData(),
	data.getData(),
	kernel.getData(),
	H, W, kH, kW);
  }
  else if (type == "valid") {
    convn_valid_kernel<<< grids, threads, 0, stream >>>(
	output.getData(),
	data.getData(),
	kernel.getData(),
	H, W, kH, kW);
  }
  else if (type == "full") {
    convn_full_kernel<<< grids, threads, 0, stream >>>(
	output.getData(),
	data.getData(),
	kernel.getData(),
	H, W, kH, kW);
  }

  CCE(hipDeviceSynchronize());
  
  return output;
}

mat xcorrn(const mat& data, const mat& kernel, string type) {
  // TODO
  return mat();
}

__global__ void downsample_kernel(float *dst, float *src, size_t scale, int H, int W) { 
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Matrix index
  int x = blockIdx.x*blockDim.x + tx;
  int y = blockIdx.y*blockDim.y + ty;

  int h = H / scale,
      w = W / scale;

  if (x >= w || y >= h)
    return;

  float sum;
  for (int i=0; i<scale; ++i) {
    for (int j=0; j<scale; ++j) {
      if ( x*scale + i < W && y*scale + j < H )
	sum += src[(x*scale + i) * H + (y*scale + j)];
    }
  }

  dst[x * h + y] = sum / (scale * scale);
}

__global__ void upsample_kernel(float *dst, float *src, size_t scale, int h, int w) { 
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Matrix index
  int x = blockIdx.x*blockDim.x + tx;
  int y = blockIdx.y*blockDim.y + ty;

  int H = h * scale,
      W = w * scale;

  if (x >= W || y >= H)
    return;

  dst[x * H + y] = src[(x / scale) * h + (y / scale)];
}

mat downsample(const mat& x, size_t scale) {
  mat output(x.getRows() / scale, x.getCols() / scale);

  ALLOCATE_GRIDS_AND_THREADS(output.getRows(), output.getCols());

  downsample_kernel<<<grids, threads>>>(
      output.getData(),
      x.getData(),
      scale,
      x.getRows(),
      x.getCols());

  CCE(hipDeviceSynchronize());

  return output;
}

mat upsample(const mat& x, size_t scale) {
  mat output(x.getRows() * scale, x.getCols() * scale);

  ALLOCATE_GRIDS_AND_THREADS(output.getRows(), output.getCols());

  upsample_kernel<<<grids, threads>>>(
      output.getData(),
      x.getData(),
      scale,
      x.getRows(),
      x.getCols());

  CCE(hipDeviceSynchronize());

  return output;
}

mat rot180(const mat& x) {
  // TODO ROTATE 180 degree (OR create another __global__ called cross_correlation
  return x;
}

/* ! \brief Sum all the elements in a matrix.
 * \fn sum_all(const device_matrix<float>& x)
 * \param x matrix x to be sum
 * return the result in host memory.
 */
float sum_all(const mat& x) {
  int r = x.getRows(),
      c = x.getCols();
  mat d_s = (mat(1, r) += 1) * x * (mat(c, 1) += 1);

  float s;
  CCE(hipMemcpy(&s, d_s.getData(), sizeof(float), hipMemcpyDeviceToHost));
  return s;
}

/* Codes for unit-testing 
 * 
 * 
 */

void plotL2normInSemilogy() {
  const float threshold = 1e-6;
  printf("N = length(L2norm);\n");
  printf("threshold = %f * ones(1, N);\n", threshold);
  printf("semilogy(1:N, L2norm, 1:N, threshold);\n");
  printf("axis([1, N, %e, %e]);\n", threshold / 100, threshold * 100);
  printf("legend('Minimum Acceptable Error', 'L2-norm');\n");
}

void test_downsample() {

  int counter = 1;

  for (int i = 0; i<20; ++i) {
    int M = rand() % 35 + 69,
	N = rand() % 43 + 28;

    mat x = rand(M, N);

    for (int scale = 2; scale < 10; ++scale) {
      mat y = downsample(x, scale);

      matlog(x);
      matlog(y);

      printf("tmp = convn(x, ones(%d) / (%d ^ 2), 'valid');\n", scale, scale);
      printf("y_gold = tmp(1:%d:end, 1:%d:end);\n", scale, scale);
      printf("delta = y - y_gold;\n");
      printf("L2norm(%d) = norm(delta(:)) / norm(y_gold(:)) / 2;\n", counter++);
    }
  }

  plotL2normInSemilogy();
}

void test_convn(string type, int N) {

// #undef matlog
// #define matlog(x) { printf(#x" = [\n"); x.print(); printf("];\n"); }

  for (int i=0; i<N; ++i) {
    int W = rand() % 50 + 5,
	H = rand() % 50 + 5,
	kW = rand() % (W-1) + 1,
	kH = rand() % (H-1) + 1;

    mat data = rand(W, H);
    mat kernel = rand(kW, kH);

    mat z = convn(data, kernel, type);
    matlog(data);
    matlog(kernel);
    matlog(z);

    printf("z_gold = convn(data, kernel, '%s');\n", type.c_str());
    printf("delta = z_gold - z;\n");
    printf("L2norm(%d) = norm(delta(:)) / norm(z_gold(:)) / 2;\n", i + 1);
  }

  plotL2normInSemilogy();
}

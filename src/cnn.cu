#include <cnn.h>
#define RED_ERROR (string("\33[31m[Error]\33[0m In function \"") \
    + __func__ + string("\" (at ") + __FILE__ + string(":") \
    + to_string(__LINE__) + string("): "))

/*! 
 * Implementation of CNN goes here.
 */

CNN::CNN(): _transforms() {

}

CNN::CNN(const string& model_fn) : _transforms() {
  this->read(model_fn);
}

CNN::~CNN() {
  for (size_t i=0; i<_transforms.size(); ++i)
    delete _transforms[i];
}

// Perform the reverse of concat
vector<mat> de_concat(const mat& concated_features, int N) {

  int batch_size = concated_features.getCols();
  vector<mat> smalls(N);

  int MAP_SIZE = concated_features.size() / N;

  SIZE s(MAP_SIZE / batch_size, batch_size);
  
  for (int i=0; i<N; ++i) {
    smalls[i].resize(s.m, s.n);
    CCE(hipMemcpy(smalls[i].getData(),
		   concated_features.getData() + i * MAP_SIZE,
	  	   sizeof(float) * MAP_SIZE,
	  	   hipMemcpyDeviceToDevice));
  }
  CCE(hipDeviceSynchronize());

  return smalls;
}

mat concat(const vector<mat>& smalls) {
  int nFeatures = smalls.size(),
      img_size  = smalls[0].getRows(),
      batchSize = smalls[0].getCols();

  mat big(img_size * nFeatures, batchSize);

  int MAP_SIZE = smalls[0].size();

  for (int i=0; i<nFeatures; ++i) {
    CCE(hipMemcpy(big.getData() + i * MAP_SIZE,
		   smalls[i].getData(),
		   sizeof(float) * MAP_SIZE,
		   hipMemcpyDeviceToDevice));
  }

  CCE(hipDeviceSynchronize());

  return big;
}

void CNN::feedForward(mat& fout, const mat& fin) {

  // First 1st layer of CNN MUST have only 1 input feature map
  vector<mat> fins;

  // Transpose the input feature (fin) so that rows = feature dimension, cols =
  // the number of data in a single batch.
  fins.push_back(~fin);

  // FIXME SubSamplingLayer does NOT need temporary buffer.
  // MAYBE just reserve those for ConvolutionalLayer.
  _houts.resize(_transforms.size());

  _transforms[0]->feedForward(_houts[0], fins);

  for (size_t i=1; i<_transforms.size(); ++i)
    _transforms[i]->feedForward(_houts[i], _houts[i-1]);

  // Concatenate
  fout = ~concat(_houts.back());
}

void CNN::backPropagate(mat& error, const mat& fin, const mat& fout,
    float learning_rate) {

  int N = _transforms.back()->getNumOutputMaps();
  vector<mat> fouts = de_concat(~fout, N),
	      errors = de_concat(~error, N);

  vector<mat> fins;
  fins.push_back(~fin);

  _transforms.back()->backPropagate(errors, _houts.back(), fouts, learning_rate);

  for (int i=_transforms.size() - 2; i >= 1; --i)
    _transforms[i]->backPropagate(errors, _houts[i-1], _houts[i], learning_rate);

  _transforms[0]->backPropagate(errors, fins, _houts[0], learning_rate);

  // Concatenate
  error = ~concat(errors);
}

void CNN::feedBackward(mat& error, const mat& delta) {
  // TODO
}

void CNN::init(const string &structure, SIZE img_size) {

  vector<string> layers = split(structure, '-');

  size_t nInputMaps = 1;

  for (size_t i=0; i<layers.size(); ++i) {

    if (layers[i].find("s") != string::npos) {
      size_t scale = str2int(layers[i].substr(0, layers[i].size() - 1));

      size_t nOutputMaps = nInputMaps;
      MIMOFeatureTransform* t = new SubSamplingLayer( nInputMaps, nOutputMaps, scale);
      t->set_input_img_size(img_size);
      _transforms.push_back(t);

      // Set the input img_size of next layer to be the output img_size of current layer.
      img_size = t->get_output_img_size();
    }
    else if (layers[i].find("x") != string::npos) {

      vector<string> dims = split(layers[i], 'x');

      size_t nOutputMaps   = str2int(dims[0]),
	     kernel_width  = str2int(dims[1]),
	     kernel_height = str2int(dims[2]);

      MIMOFeatureTransform* t =
	new ConvolutionalLayer( nInputMaps, nOutputMaps, kernel_height, kernel_width);

      t->set_input_img_size(img_size);

      _transforms.push_back(t);

      // Set the input img_size of next layer to be the output img_size of current layer.
      img_size = t->get_output_img_size();
      nInputMaps = nOutputMaps;
    }
    else
      throw std::runtime_error("\33[31m[Error]\33[0m No such type of layer. \""
	  + layers[i] + "\". Only convolutional/sub-sampling layer are allowed");

  }
}

void CNN::read(const string &fn) {
  // TODO

}

void CNN::save(const string &fn) const {
  // TODO
}

void CNN::status() const {

  printf("+--------------+---------------+--------------+---------------+\n");
  printf("| # input maps | # output maps | kernel width | kernel height |\n");

  for (size_t i=0; i<_transforms.size(); ++i)
    _transforms[i]->status();

  printf("+-------------------------------------------------------------+\n");
}

/*! 
 * Implementation of ConvolutionalLayer goes here.
 */
ConvolutionalLayer::ConvolutionalLayer(size_t n, size_t m, int h, int w)
  : MIMOFeatureTransform(n, m) {
  if (w == -1)
    w = h;

  assert(n > 0 && m > 0 && h > 0 && w > 0);

  printf("Initializing %lu x %lu kernels of size %d x %d\n", n, m, h, w);
  _kernels.resize(n);
  for (size_t i=0; i<n; ++i)
    _kernels[i].assign(m, rand(h, w));

  _bias.resize(m);
  for (size_t j=0; j<m; ++j)
    _bias[j] = 0;
}

/* TODO If every element in fins is a single feature map, then only a data can
 *      be fed forward through this function.
 *      NOTE that fins.size()  == # of input feature maps
 *                             != # of data in a batch
 *
 *	To feed forward a whole batch in a single function:
 *                fins.size()  == # of input feature maps
 *		  fins[i].rows == map.rows x map.cols
 *		  fins[i].cols == # of data
 *
 *	That is fins.size() is still the # of input feature maps (, which is
 *      always inevitable). However, in the i-th element of fins (i.e. fins[i])
 *	, there're multiple input feature maps comes from multiple training data.
 * */

void ConvolutionalLayer::feedForward(vector<mat>& fouts, const vector<mat>& fins) {

  size_t nInputs  = getNumInputMaps(),
	 nOutputs = getNumOutputMaps();

  if (fins.size() != nInputs)
    throw std::runtime_error(RED_ERROR + "Number of inputs maps ( = "
	+ to_string(fins.size()) + ") does not match number of kernels ( = "
	+ to_string(nInputs) + ").");

  size_t batch_size = fins[0].getCols();

  vector<vector<mat> > iImgs(nInputs), oImgs(nOutputs);

  SIZE s = get_output_img_size();

  for (size_t i=0; i<nInputs; ++i)
    iImgs[i] = reshapeVectors2Images(fins[i], _input_img_size);

  // Allocate memory and initialize with value 0
  for (size_t j=0; j<nOutputs; ++j) {
    oImgs[j].resize(batch_size);

    for (size_t k=0; k<batch_size; ++k)
      oImgs[j][k].resize(s.m, s.n, 0);
  }

  for (size_t k=0; k<batch_size; ++k) {
    for (size_t j=0; j<nOutputs; ++j) {
      for (size_t i=0; i<nInputs; ++i)
	oImgs[j][k] += convn(iImgs[i][k], _kernels[i][j], "valid_shm");
      oImgs[j][k] += _bias[j];
    }
  }

  if (fouts.size() != nOutputs)
    fouts.resize(nOutputs);

  for (size_t j=0; j<nOutputs; ++j)
    fouts[j] = sigmoid(reshapeImages2Vectors(oImgs[j]));

}

void ConvolutionalLayer::feedBackward(
    vector<mat>& errors, const vector<mat>& deltas) {

  // Since nInputs == nOutputs for subsampling layer, I just use N.
  size_t nInputs = getNumInputMaps(),
	 nOutputs = getNumOutputMaps();

  SIZE s = this->get_input_img_size();
  size_t batch_size = deltas[0].getCols();

  /* ... DEBUG CODES BEGIN HERE ... */
  /*errors.resize(nInputs);
  for (int i=0; i<nInputs; ++i)
    errors[i].resize(s.m * s.n, batch_size);
  return;*/
  /* ... DEBUG CODES  END  HERE ... */

  vector<vector<mat> > oImgs(nOutputs), iImgs(nInputs);
  for (size_t j=0; j<nOutputs; ++j)
    oImgs[j] = reshapeVectors2Images(deltas[j], this->get_output_img_size());


  for (size_t i=0; i<nInputs; ++i)
    iImgs[i].resize(batch_size);

  // FIXME beware that upsample may NOT be able to get back to original size
  for (size_t k=0; k<batch_size; ++k) {
    for (size_t i=0; i<nInputs; ++i) {
      iImgs[i][k].resize(s.m, s.n, 0);
      for (size_t j=0; j<nOutputs; ++j) {
	iImgs[i][k] += convn(oImgs[j][k], rot180(_kernels[i][j]), "full");
      }
    }
  }

  if (errors.size() != nInputs)
    errors.resize(nInputs);

  for (size_t i=0; i<nInputs; ++i)
    errors[i] = reshapeImages2Vectors(iImgs[i]);

}

// NOTE: in MATLAB
// xcorr2 stands for 2D cross-correlation
// (I don't know why MATLAB does not have "xcorrn" for n-dimensional xcorr)
// The following operation are theoretically equivalent:
// (with only some trivial numerical error)
// (1)  convn(x, rot180(h)) == xcorr2(x, h)
//     xcorr2(x, rot180(h)) ==  convn(x, h)
// (2) convn(rot180(x), h) == rot180(convn(x, rot180(h)))
//     ^
//     |_____ which is obviously faster

void ConvolutionalLayer::backPropagate(vector<mat>& errors, const vector<mat>& fins,
    const vector<mat>& fouts, float learning_rate) {

  size_t nInputs = getNumInputMaps(),
	 nOutputs = getNumOutputMaps();

  size_t batch_size = fins[0].getCols();

  // In the following codes, the iteration index i and j stands for
  // i : # of input  features. i = 0 ~ nInputs - 1 
  // j : # of output features. j = 0 ~ nOutputs - 1

  vector<mat> deltas(nOutputs);
  for (size_t j=0; j<nOutputs; ++j)
    deltas[j] = fouts[j] & ( 1.0f - fouts[j] ) & errors[j];

  // iImgs represents the input images.
  // oImgs represents the output images. (Before sigmoid or any other activation function)
  vector<vector<mat> > iImgs(nInputs), oImgs(nOutputs);

  for (size_t i=0; i<nInputs; ++i)
    iImgs[i] = reshapeVectors2Images(fins[i], _input_img_size);

  for (size_t j=0; j<nOutputs; ++j)
    oImgs[j] = reshapeVectors2Images(deltas[j], this->get_output_img_size());

  assert(learning_rate > 0);
  float lr = learning_rate / batch_size;

  // Update kernels with learning rate
  for (size_t k=0; k<batch_size; ++k) {
    for (size_t j=0; j<nOutputs; ++j) {
      for (size_t i=0; i<nInputs; ++i)
	_kernels[i][j] -= convn(rot180(iImgs[i][k]), oImgs[j][k], "valid") * lr;

      _bias[j] -= sum_all(oImgs[j][k]) * lr;
    }
  }
  
  this->feedBackward(errors, deltas);
}

void ConvolutionalLayer::status() const {

  printf("+--------------+---------------+--------------+---------------+\n");
  printf("|      %-5lu   |       %-5lu   |      %-5lu   |       %-5lu   |\n",
      getNumInputMaps(), getNumOutputMaps(), getKernelWidth(), getKernelHeight());
}

size_t ConvolutionalLayer::getKernelWidth() const {
  return _kernels[0][0].getCols();
}

size_t ConvolutionalLayer::getKernelHeight() const {
  return _kernels[0][0].getRows();
}

/*size_t ConvolutionalLayer::getNumInputMaps() const {
  return _kernels.size();
}

size_t ConvolutionalLayer::getNumOutputMaps() const {
  return _kernels[0].size();
}*/

SubSamplingLayer::SubSamplingLayer(size_t m, size_t n, size_t scale)
  : MIMOFeatureTransform(m, n), _scale(scale) {
}

void SubSamplingLayer::status() const {
  printf("+-------------------------------------------------------------+\n");
  printf("|                Sub-Sampling Factor = %-4lu                   |\n", _scale);
}
  
size_t SubSamplingLayer::getScale() const {
  return _scale;
}

void SubSamplingLayer::feedForward(vector<mat>& fouts, const vector<mat>& fins) {

  // Since nInputs == nOutputs for subsampling layer, I just use N.
  size_t N = fins.size();
  size_t batch_size = fins[0].getCols();

  vector<vector<mat> > iImgs(N), oImgs(N);
  for (size_t i=0; i<N; ++i)
    iImgs[i] = reshapeVectors2Images(fins[i], _input_img_size);

  for (size_t i=0; i<N; ++i) {
    oImgs[i].resize(batch_size);
    for (size_t k=0; k<batch_size; ++k)
      oImgs[i][k] = downsample(iImgs[i][k], _scale);
  }

  if (fouts.size() != N)
    fouts.resize(N);

  for (size_t j=0; j<N; ++j)
    fouts[j] = reshapeImages2Vectors(oImgs[j]);
}

void SubSamplingLayer::feedBackward(
    vector<mat>& errors, const vector<mat>& deltas) {

  // Since nInputs == nOutputs for subsampling layer, I just use N.
  size_t N = deltas.size();
  size_t batch_size = deltas[0].getCols();

  /* ... DEBUG CODES BEGIN HERE ... */
  /*errors.resize(N);
  for (int i=0; i<N; ++i)
    errors[i].resize(_input_img_size.m * _input_img_size.n, batch_size);
  return;*/
  /* ... DEBUG CODES  END  HERE ... */

  vector<vector<mat> > oImgs(N), iImgs(N);
  for (size_t i=0; i<N; ++i)
    oImgs[i] = reshapeVectors2Images(deltas[i], this->get_output_img_size());

  // FIXME beware that upsample may NOT be able to get back to original size
  for (size_t i=0; i<N; ++i) {
    iImgs[i].resize(batch_size);
    for (size_t k=0; k<batch_size; ++k)
      iImgs[i][k] = upsample(oImgs[i][k], _scale, _input_img_size);
  }

  if (errors.size() != N)
    errors.resize(N);

  for (size_t j=0; j<N; ++j)
    errors[j] = reshapeImages2Vectors(iImgs[j]);
}

void SubSamplingLayer::backPropagate(vector<mat>& errors, const vector<mat>& fins,
    const vector<mat>& fouts, float learning_rate) {

  // Copy errors element by element to deltas
  vector<mat> deltas(errors);

  this->feedBackward(errors, deltas);
}

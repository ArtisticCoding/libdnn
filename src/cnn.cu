#include "hip/hip_runtime.h"
#include <cnn.h>

void plotL2normInSemilogy() {
  const float threshold = 1e-6;
  printf("N = length(L2norm);\n");
  printf("threshold = %f * ones(1, N);\n", threshold);
  printf("semilogy(1:N, L2norm, 1:N, threshold);\n");
  printf("axis([1, N, %e, %e]);\n", threshold / 100, threshold * 100);
  printf("legend('Minimum Acceptable Error', 'L2-norm');\n");
}

mat rand(int m, int n) {
  mat x(m, n);
  ext::rand(x);
  return x;
}

void test_convn(string type, int N) {

// #undef matlog
// #define matlog(x) { printf(#x" = [\n"); x.print(); printf("];\n"); }

  for (int i=0; i<N; ++i) {
    int W = rand() % 50 + 5,
	H = rand() % 50 + 5,
	kW = rand() % (W-1) + 1,
	kH = rand() % (H-1) + 1;

    mat data = rand(W, H);
    mat kernel = rand(kW, kH);

    mat z = convn(data, kernel, type);
    matlog(data);
    matlog(kernel);
    matlog(z);

    printf("z_gold = convn(data, kernel, '%s');\n", type.c_str());
    printf("delta = z_gold - z;\n");
    printf("L2norm(%d) = norm(delta(:)) / norm(z_gold(:)) / 2;\n", i + 1);
  }

  plotL2normInSemilogy();
}

__global__ void convn_valid_kernel(float *output, float *data, float *kernel, int H, int W, int kH, int kW) { 
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Matrix index
  int x = blockIdx.x*blockDim.x + tx;
  int y = blockIdx.y*blockDim.y + ty;

  // vH, vW stands for valid H and valid W
  const int vH = H - kH + 1,
	    vW = W - kW + 1;

  if (x >= vW || y >= vH)
    return;

  x += kW - 1;
  y += kH - 1;

  float sum = 0; 
  for (int i = 0; i < kW; ++i)
    for(int j = 0; j < kH; ++j)
      sum += kernel[ i * kH + j ] * data[ (x - i) * H + (y - j) ]; 

  x -= kW - 1;
  y -= kH - 1;

  output[ x * vH + y ] = sum;
} 

__global__ void convn_same_kernel(float *output, float *data, float *kernel, int H, int W, int kH, int kW) { 
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Matrix index
  int x = blockIdx.x*blockDim.x + tx;
  int y = blockIdx.y*blockDim.y + ty;

  if (x >= W || y >= H)
    return;

  const int i0 = kW / 2, j0 = kH / 2;

  float sum = 0; 
  for (int i = 0; i < kW; ++i) {
    for(int j = 0; j < kH; ++j) {
      int ii = x - i + i0, jj = y - j + j0;

      if ( ii < 0 || ii >= W || jj < 0 || jj >= H )
	continue;

      sum += kernel[ i * kH + j ] * data[ ii * H + jj ]; 
    }
  }

  output[x * H + y] = sum;
} 


__global__ void convn_full_kernel(float *output, float *data, float *kernel, int H, int W, int kH, int kW) { 
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Matrix index
  int x = blockIdx.x*blockDim.x + tx;
  int y = blockIdx.y*blockDim.y + ty;

  // fH, fW stands for full H and full W
  const int fH = H + kH - 1,
	    fW = W + kW - 1;

  if (x >= fW || y >= fH)
    return;

  float sum = 0; 
  for (int i = 0; i < kW; ++i) {
    for(int j = 0; j < kH; ++j) {
      int ii = x - i, jj = y - j;

      if ( ii < 0 || ii >= W || jj < 0 || jj >= H )
	continue;

      sum += kernel[ i * kH + j ] * data[ ii * H + jj ]; 
    }
  }

  output[ x * fH + y ] = sum;
}

mat convn(const mat& data, const mat& kernel, string type) {

  const size_t N = 32;
  dim3 threads(N, N);
  dim3 grid;
  
  int H = data.getRows(),
      W = data.getCols(),
      kH = kernel.getRows(),
      kW = kernel.getCols();

  mat output;
  
  if (type == "same")
    output.resize(H, W);
  else if (type == "valid") {
    int a = max(H - kH + 1, 0),
	b = max(W - kW + 1, 0);

    if (a == 0 || b == 0)
      return mat();

    output.resize(a, b);
  }
  else if (type == "full")
    output.resize(H + kH - 1, W + kW - 1);
  else
    throw std::runtime_error("No such type of convolution");

  grid.x = (unsigned int) ceil((float) output.getCols() / N);
  grid.y = (unsigned int) ceil((float) output.getRows() / N);


  if (type == "same") {
    convn_same_kernel<<<grid, threads>>>(
	output.getData(),
	data.getData(),
	kernel.getData(),
	H, W, kH, kW);
  }
  else if (type == "valid") {
    convn_valid_kernel<<<grid, threads>>>(
	output.getData(),
	data.getData(),
	kernel.getData(),
	H, W, kH, kW);
  }
  else if (type == "full") {
    convn_full_kernel<<<grid, threads>>>(
	output.getData(),
	data.getData(),
	kernel.getData(),
	H, W, kH, kW);
  }

  CCE(hipDeviceSynchronize());
  
  return output;
}

__global__ void downsample_kernel(float *dst, float *src, size_t scale, int H, int W) { 
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Matrix index
  int x = blockIdx.x*blockDim.x + tx;
  int y = blockIdx.y*blockDim.y + ty;

  int h = H / scale,
      w = W / scale;

  if (x >= w || y >= h)
    return;

  float sum;
  for (int i=0; i<scale; ++i) {
    for (int j=0; j<scale; ++j) {
      if ( x*scale + i < W && y*scale + j < H )
	sum += src[(x*scale + i) * H + (y*scale + j)];
    }
  }

  dst[x * h + y] = sum / (scale * scale);
}


mat downsample(const mat& x, size_t scale) {
  mat output(x.getRows() / scale, x.getCols() / scale);

  const size_t N = 32;
  dim3 threads(N, N);
  dim3 grid;
  
  grid.x = (unsigned int) ceil((float) output.getCols() / N);
  grid.y = (unsigned int) ceil((float) output.getRows() / N);

  downsample_kernel<<<grid, threads>>>(
      output.getData(),
      x.getData(),
      scale,
      x.getRows(),
      x.getCols());

  CCE(hipDeviceSynchronize());

  return output;
}

void test_downsample() {

  int counter = 1;

  for (int i = 0; i<20; ++i) {
    int M = rand() % 35 + 69,
	N = rand() % 43 + 28;

    mat x = rand(M, N);

    for (int scale = 2; scale < 10; ++scale) {
      mat y = downsample(x, scale);

      matlog(x);
      matlog(y);

      printf("tmp = convn(x, ones(%d) / (%d ^ 2), 'valid');\n", scale, scale);
      printf("y_gold = tmp(1:%d:end, 1:%d:end);\n", scale, scale);
      printf("delta = y - y_gold;\n");
      printf("L2norm(%d) = norm(delta(:)) / norm(y_gold(:)) / 2;\n", counter++);
    }
  }

  plotL2normInSemilogy();
}


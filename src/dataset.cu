#include "hip/hip_runtime.h"
#include <dataset.h>
#include <dnn-utility.h>

DataSet::DataSet() {
}

DataSet::DataSet(const string &fn, bool rescale) {

  read(fn, rescale);

  _y = getStandardLabels();
  _prob = label2PosteriorProb(_y);
}

mat DataSet::getStandardLabels() {
  assert(_y.getCols() == 1);

  size_t N = _y.getRows();
  float* hy = new float[N];
  CCE(hipMemcpy(hy, _y.getData(), sizeof(float) * N, hipMemcpyDeviceToHost));

  // Replace labels to 1, 2, 3, N, using mapping
  map<int, int> classes = getLabelMapping(_y);
  for (size_t i=0; i<N; ++i)
    hy[i] = classes[hy[i]];

  mat sLabels(hy, N, 1);
  delete [] hy;

  return sLabels;
}

size_t DataSet::getInputDimension() const {
  // FIXME the input dimension shouldn't be so unclear
  return _X.getCols() - 1;
}

size_t DataSet::getOutputDimension() const {
  return _prob.getCols();
}

void DataSet::rescaleFeature(float* data, size_t rows, size_t cols, float lower, float upper) {
  for (size_t i=0; i<rows; ++i) {
    float min = data[i],
	  max = data[i];

    for (size_t j=0; j<cols; ++j) {
      float x = data[j*rows + i];
      if (x > max) max = x;
      if (x < min) min = x;
    }

    if (max == min) {
      for (size_t j=0; j<cols; ++j)
	data[j*rows + i] = upper;
      continue;
    }

    float ratio = (upper - lower) / (max - min);
    for (size_t j=0; j<cols; ++j)
      data[j*rows + i] = (data[j*rows + i] - min) * ratio + lower;
  }
}

void DataSet::read(const string &fn, bool rescale) {
  ifstream fin(fn.c_str());

  bool isSparse = isFileSparse(fn);

  size_t cols = isSparse ? findMaxDimension(fin) : findDimension(fin);
  size_t rows = getLineNumber(fin);
  float* data = new float[rows * cols];
  float* labels = new float[rows];

  memset(data, 0, sizeof(float) * rows * cols);

  if (isSparse)
    readSparseFeature(fin, data, labels, rows, cols);
  else
    readDenseFeature(fin, data, labels, rows, cols);

  fin.close();

  // --------------------------------------
  if (rescale) {
    printf("\33[33m[Info]\33[0m rescale each feature to [0, 1]\n");
    rescaleFeature(data, rows, cols);
  }

  _X = mat(data, rows, cols);
  _X.reserve(rows * (cols + 1));
  _X.resize(rows, cols + 1);
  fillLastColumnWith(_X, (float) 1.0);

  _y = mat(labels, rows, 1);
  // --------------------------------------

  delete [] data;
  delete [] labels;
}

void DataSet::readSparseFeature(ifstream& fin, float* data, float* labels, size_t rows, size_t cols) {

  string line, token;
  size_t i = 0;
  while (std::getline(fin, line)) {
    stringstream ss(line);
  
    ss >> token;
    labels[i] = str2float(token);

    while (ss >> token) {
      size_t pos = token.find(':');
      if (pos == string::npos)
	continue;

      size_t j = str2float(token.substr(0, pos)) - 1;
      float value = str2float(token.substr(pos + 1));
      
      data[j * rows + i] = value;
    }
    ++i;
  }
}

void DataSet::readDenseFeature(ifstream& fin, float* data, float* labels, size_t rows, size_t cols) {

  string line, token;
  size_t i = 0;
  while (std::getline(fin, line)) {
    stringstream ss(line);
  
    ss >> token;
    labels[i] = str2float(token);

    size_t j = 0;
    while (ss >> token)
      data[(j++) * rows + i] = str2float(token);
    ++i;
  }
}

void DataSet::showSummary() const {
  size_t input_dim  = _X.getCols();
  size_t nData	    = _X.getRows();
  size_t nClasses   = _prob.getCols();

  printf("+--------------------------------+-----------+\n");
  printf("| Number of classes              | %9lu |\n", nClasses);
  printf("| Number of input feature (data) | %9lu |\n", nData);
  printf("| Dimension of  input feature    | %9lu |\n", input_dim);
  printf("+--------------------------------+-----------+\n");

}

size_t DataSet::getClassNumber() const {
  thrust::device_ptr<float> dptr(_y.getData());
  thrust::host_vector<float> y(dptr, dptr + _y.size());

  map<float, bool> classes;
  for (size_t i=0; i<_y.size(); ++i)
    classes[y[i]] = true;

  return classes.size();
}

void DataSet::shuffleFeature() {

  float *h_X = new float[_X.size()],
	*h_y = new float[_y.size()];

  CCE(hipMemcpy(h_X, _X.getData(), sizeof(float) * _X.size(), hipMemcpyDeviceToHost));
  CCE(hipMemcpy(h_y, _y.getData(), sizeof(float) * _y.size(), hipMemcpyDeviceToHost));

  shuffleFeature(h_X, h_y, _X.getRows(), _X.getCols());

  CCE(hipMemcpy(_X.getData(), h_X, sizeof(float) * _X.size(), hipMemcpyHostToDevice));
  CCE(hipMemcpy(_y.getData(), h_y, sizeof(float) * _y.size(), hipMemcpyHostToDevice));

  _prob = label2PosteriorProb(_y);

  delete [] h_X;
  delete [] h_y;
}

bool isFileSparse(string train_fn) {
  ifstream fin(train_fn.c_str());
  string line;
  std::getline(fin, line);
  return line.find(':') != string::npos;
}

size_t getLineNumber(ifstream& fin) {
  int previous_pos = fin.tellg();
  string a;
  size_t n = 0;
  while(std::getline(fin, a) && ++n);
  fin.clear();
  fin.seekg(previous_pos);
  return n;
}

size_t findMaxDimension(ifstream& fin) {
  int previous_pos = fin.tellg();

  string token;
  size_t maxDimension = 0;
  while (fin >> token) {
    size_t pos = token.find(':');
    if (pos == string::npos)
      continue;

    size_t dim = atoi(token.substr(0, pos).c_str());
    if (dim > maxDimension)
      maxDimension = dim;
  }

  fin.clear();
  fin.seekg(previous_pos);

  return maxDimension;
}

size_t findDimension(ifstream& fin) {

  size_t dim = 0;

  int previous_pos = fin.tellg();

  string line;
  std::getline(fin, line);
  stringstream ss(line);

  // First token is class label
  string token;
  ss >> token;

  while (ss >> token)
    ++dim;
  
  fin.clear();
  fin.seekg(previous_pos);

  return dim;
}

void DataSet::shuffleFeature(float* const data, float* const labels, int rows, int cols) {

  std::vector<size_t> perm = randperm(rows);

  float* tmp_data = new float[rows*cols];
  float* tmp_labels = new float[rows];

  memcpy(tmp_data, data, sizeof(float) * rows * cols);
  memcpy(tmp_labels, labels, sizeof(float) * rows);

  for (size_t i=0; i<rows; ++i) {
    size_t from = i;
    size_t to = perm[i];

    for (size_t j=0; j<cols; ++j)
      data[j * rows + to] = tmp_data[j * rows + from];

    labels[to] = tmp_labels[from];
  }

  delete [] tmp_data;
  delete [] tmp_labels;
}

mat& DataSet::getX() {
  return _X;
}

const mat& DataSet::getX() const {
  return _X;
}

mat& DataSet::getY() {
  return _y;
}

const mat& DataSet::getY() const {
  return _y;
}

mat& DataSet::getProb() {
  return _prob;
}

const mat& DataSet::getProb() const {
  return _prob;
}

void splitIntoTrainingAndValidationSet(
    DataSet& train, DataSet& valid,
    DataSet& data, int ratio) {

  size_t rows = data.getX().getRows(),
	 inputDim = data.getX().getCols(),
	 outputDim = data.getProb().getCols();
  
  float *h_X = new float[rows*inputDim],
	*h_y = new float[rows],
        *h_prob = new float[rows*outputDim];

  CCE(hipMemcpy(h_X, data.getX().getData(), sizeof(float) * data.getX().size(), hipMemcpyDeviceToHost));
  CCE(hipMemcpy(h_y, data.getY().getData(), sizeof(float) * data.getY().size(), hipMemcpyDeviceToHost));
  CCE(hipMemcpy(h_prob, data.getProb().getData(), sizeof(float) * data.getProb().size(), hipMemcpyDeviceToHost));

  float* h_trainX, *h_trainY, *h_trainProb, *h_validX, *h_validY, *h_validProb;
  size_t nTrain, nValid;
  splitIntoTrainingAndValidationSet(
      h_trainX, h_trainProb, h_trainY, nTrain,
      h_validX, h_validProb, h_validY, nValid,
      ratio,
      h_X, h_prob, h_y,
      rows, inputDim, outputDim);

  train.getX()    = mat(h_trainX   , nTrain, inputDim );
  train.getProb() = mat(h_trainProb, nTrain, outputDim);
  train.getY()    = mat(h_trainY   , nTrain, 1        );

  valid.getX()    = mat(h_validX   , nValid, inputDim );
  valid.getProb() = mat(h_validProb, nValid, outputDim);
  valid.getY()    = mat(h_validY   , nValid, 1	      );

  delete [] h_X;
  delete [] h_prob;
  delete [] h_y;

  delete [] h_trainX;
  delete [] h_trainY;
  delete [] h_trainProb;

  delete [] h_validX;
  delete [] h_validY;
  delete [] h_validProb;
}

void splitIntoTrainingAndValidationSet(
    float* &trainX, float* &trainProb, float* &trainY, size_t& nTrain,
    float* &validX, float* &validProb, float* &validY, size_t& nValid,
    int ratio, /* ratio of training / validation */
    const float* const data, const float* const prob, const float* const labels,
    int rows, int inputDim, int outputDim) {

  nValid = rows / ratio;
  nTrain = rows - nValid;
  printf("| nTrain                         | %9lu |\n", nTrain);
  printf("| nValid                         | %9lu |\n", nValid);

  trainX    = new float[nTrain * inputDim];
  trainProb = new float[nTrain * outputDim];
  trainY    = new float[nTrain];

  validX    = new float[nValid * inputDim];
  validProb = new float[nValid * outputDim];
  validY    = new float[nValid];

  for (size_t i=0; i<nTrain; ++i) {
    for (size_t j=0; j<inputDim; ++j)
      trainX[j * nTrain + i] = data[j * rows + i];
    for (size_t j=0; j<outputDim; ++j)
      trainProb[j * nTrain + i] = prob[j * rows + i];
    trainY[i] = labels[i];
  }

  for (size_t i=0; i<nValid; ++i) {
    for (size_t j=0; j<inputDim; ++j)
      validX[j * nValid + i] = data[j * rows + i + nTrain];
    for (size_t j=0; j<outputDim; ++j)
      validProb[j * nValid + i] = prob[j * rows + i + nTrain];
    validY[i] = labels[i + nTrain];
  }
}

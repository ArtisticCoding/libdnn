#include "hip/hip_runtime.h"
#include <dataset.h>
#include <dnn-utility.h>

DataSet::DataSet() {
}

DataSet::DataSet(const string &fn, bool rescale) {

  read(fn, rescale);

  convertToStandardLabels();
  label2PosteriorProb();
}

void DataSet::convertToStandardLabels() {
  assert(_hy.getCols() == 1);

  // Replace labels to 1, 2, 3, N, using mapping
  map<int, int> classes = getLabelMapping(_hy);
  for (size_t i=0; i<_hy.getRows(); ++i)
    _hy.getData()[i] = classes[_hy.getData()[i]];
}

size_t DataSet::getInputDimension() const {
  // FIXME the input dimension shouldn't be so unclear
  return _hx.getCols() - 1;
}

size_t DataSet::getOutputDimension() const {
  return _hprob.getCols();
}

void DataSet::rescaleFeature(float lower, float upper) {

  size_t rows = _hx.getRows(),
	 cols = _hx.getCols();

  for (size_t i=0; i<rows; ++i) {
    float min = _hx.getData()[i],
	  max = _hx.getData()[i];

    for (size_t j=0; j<cols; ++j) {
      float x = _hx(i, j);
      if (x > max) max = x;
      if (x < min) min = x;
    }

    if (max == min) {
      for (size_t j=0; j<cols; ++j)
	_hx(i, j) = upper;
      continue;
    }

    float ratio = (upper - lower) / (max - min);
    for (size_t j=0; j<cols; ++j)
      _hx(i, j) = (_hx(i, j) - min) * ratio + lower;
  }
}

void DataSet::read(const string &fn, bool rescale) {
  ifstream fin(fn.c_str());

  bool isSparse = isFileSparse(fn);

  size_t cols = isSparse ? findMaxDimension(fin) : findDimension(fin);
  size_t rows = getLineNumber(fin);

  _hx.resize(rows, cols);
  _hy.resize(rows, 1);

  if (isSparse)
    readSparseFeature(fin);
  else
    readDenseFeature(fin);

  fin.close();

  // --------------------------------------
  if (rescale) {
    printf("\33[33m[Info]\33[0m rescale each feature to [0, 1]\n");
    rescaleFeature();
  }

  _hx.reserve(rows * (cols + 1));
  _hx.resize(rows, cols + 1);

  float* lastColumn = _hx.getData() + rows * cols;
  std::fill(lastColumn, lastColumn + rows, 1.0f);

  // --------------------------------------
}

void DataSet::readSparseFeature(ifstream& fin) {

  size_t rows = _hx.getRows(),
	 cols = _hx.getCols();

  string line, token;
  size_t i = 0;
  while (std::getline(fin, line)) {
    stringstream ss(line);
  
    ss >> token;
    _hy.getData()[i] = str2float(token);

    while (ss >> token) {
      size_t pos = token.find(':');
      if (pos == string::npos)
	continue;

      size_t j = str2float(token.substr(0, pos)) - 1;
      float value = str2float(token.substr(pos + 1));
      
      _hx.getData()[j * rows + i] = value;
    }
    ++i;
  }
}

void DataSet::readDenseFeature(ifstream& fin) {
  
  size_t rows = _hx.getRows(),
	 cols = _hx.getCols();

  string line, token;
  size_t i = 0;
  while (std::getline(fin, line)) {
    stringstream ss(line);
  
    ss >> token;
    _hy.getData()[i] = str2float(token);

    size_t j = 0;
    while (ss >> token)
      _hx.getData()[(j++) * rows + i] = str2float(token);
    ++i;
  }
}

void DataSet::showSummary() const {
  size_t input_dim  = _hx.getCols();
  size_t nData	    = _hx.getRows();
  size_t nClasses   = _hprob.getCols();

  printf("+--------------------------------+-----------+\n");
  printf("| Number of classes              | %9lu |\n", nClasses);
  printf("| Number of input feature (data) | %9lu |\n", nData);
  printf("| Dimension of  input feature    | %9lu |\n", input_dim);
  printf("+--------------------------------+-----------+\n");

}

size_t DataSet::getClassNumber() const {
  return getLabelMapping(_hy).size();
}

void DataSet::label2PosteriorProb() {
  
  map<int, int> classes = getLabelMapping(_hy);
  size_t nClasses = classes.size();

  // Convert labels to posterior probabilities
  _hprob.resize(_hy.getRows(), nClasses);
  _hprob.fillwith(0);

  for (size_t i=0; i<_hprob.getRows(); ++i)
    _hprob(i, (_hy[i] - 1)) = 1;
}

bool isFileSparse(string train_fn) {
  ifstream fin(train_fn.c_str());
  string line;
  std::getline(fin, line);
  return line.find(':') != string::npos;
}

size_t getLineNumber(ifstream& fin) {
  int previous_pos = fin.tellg();
  string a;
  size_t n = 0;
  while(std::getline(fin, a) && ++n);
  fin.clear();
  fin.seekg(previous_pos);
  return n;
}

size_t findMaxDimension(ifstream& fin) {
  int previous_pos = fin.tellg();

  string token;
  size_t maxDimension = 0;
  while (fin >> token) {
    size_t pos = token.find(':');
    if (pos == string::npos)
      continue;

    size_t dim = atoi(token.substr(0, pos).c_str());
    if (dim > maxDimension)
      maxDimension = dim;
  }

  fin.clear();
  fin.seekg(previous_pos);

  return maxDimension;
}

size_t findDimension(ifstream& fin) {

  size_t dim = 0;

  int previous_pos = fin.tellg();

  string line;
  std::getline(fin, line);
  stringstream ss(line);

  // First token is class label
  string token;
  ss >> token;

  while (ss >> token)
    ++dim;
  
  fin.clear();
  fin.seekg(previous_pos);

  return dim;
}

void DataSet::shuffleFeature() {

  size_t rows = _hx.getRows(),
	 cols = _hx.getCols();

  std::vector<size_t> perm = randperm(rows);

  hmat x(_hx), y(_hy);

  for (size_t i=0; i<rows; ++i) {
    for (size_t j=0; j<cols; ++j)
      _hx(perm[i], j) = x (i, j);
    _hy[perm[i]] = y[i];
  }

  label2PosteriorProb();
}

bool DataSet::isLabeled() const {
  return getLabelMapping(_hy).size() > 1;
}

mat DataSet::getX() const {
  return mat(_hx.getData(), _hx.getRows(), _hx.getCols());
}

mat DataSet::getY() const {
  return mat(_hy.getData(), _hy.getRows(), _hy.getCols());
}

mat DataSet::getProb() const {
  return mat(_hprob.getData(), _hprob.getRows(), _hprob.getCols());
}

void DataSet::splitIntoTrainAndValidSet(DataSet& train, DataSet& valid, int ratio) {

  size_t rows = _hx.getRows(),
	 inputDim = _hx.getCols(),
	 outputDim = _hprob.getCols();
  
  size_t nValid = rows / ratio,
	 nTrain = rows - nValid;

  printf("| nTrain                         | %9lu |\n", nTrain);
  printf("| nValid                         | %9lu |\n", nValid);

  // Copy data to training set
  train._hx.resize(nTrain, inputDim);
  train._hy.resize(nTrain, 1);
  train._hprob.resize(nTrain, outputDim);

  for (size_t i=0; i<nTrain; ++i) {
    for (size_t j=0; j<inputDim; ++j)
      train._hx(i, j) = _hx(i, j);

    for (size_t j=0; j<outputDim; ++j)
      train._hprob(i, j) = _hprob(i, j);

    train._hy[i] = _hy[i];
  }

  // Copy data to validation set
  valid._hx.resize(nValid, inputDim);
  valid._hy.resize(nValid, 1);
  valid._hprob.resize(nValid, outputDim);

  for (size_t i=0; i<nValid; ++i) {
    for (size_t j=0; j<inputDim; ++j)
      valid._hx(i, j) = _hx[j * rows + i + nTrain];

    for (size_t j=0; j<outputDim; ++j)
      valid._hprob(i, j) = _hprob[j * rows + i + nTrain];

    valid._hy[i] = _hy[i + nTrain];
  }
}
